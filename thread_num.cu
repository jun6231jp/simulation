#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include <time.h>
#include <math.h>

#define Grid_x 256
#define Grid_y 256
#define Grid_z 2
#define Block_x 32
#define Block_y 32
#define Block_z 1

__global__ void thread_num( unsigned long int  *device_result );

/* timer */
int timer(void){
  time_t now = time(NULL);
  struct tm *pnow = localtime(&now);
  char buff[128]="";
  sprintf(buff,"%d:%d:%d",pnow->tm_hour,pnow->tm_min,pnow->tm_sec);
  printf("%s\n",buff);
  return 0;
}

int main(int argc, char** argv){
  FILE *outputfile;
  outputfile = fopen("./thread_num.txt", "w");
  if (outputfile == NULL) {
    printf("cannot open\n");
    exit(1);
  }
  timer();

  /*ブロックサイズとグリッドサイズの設定*/
  dim3 grid(Grid_x,Grid_y,Grid_z);
  dim3 block(Block_x,Block_y,Block_z);

  /*ホスト側の変数設定*/
  unsigned long int  thread_size = (Grid_x * Grid_y * Grid_z) * (Block_x * Block_y * Block_z);

  unsigned long int  *host_result;
  
  /*デバイス側の変数設定*/
  unsigned long int  *device_result;

  /*デバイスメモリ領域の確保*/
  checkCudaErrors(hipMalloc((void**)&device_result, thread_size * sizeof(unsigned long int )));
  
  /*タイマーを作成して計測開始*/
  hipEvent_t start;
  hipEvent_t stop;
  checkCudaErrors(hipEventCreate(&start));
  checkCudaErrors(hipEventCreate(&stop));
  checkCudaErrors(hipEventRecord(start, NULL));
  
  /*カーネルの起動*/
  thread_num<<<grid , block>>>(device_result);
  hipDeviceSynchronize();
  
  /*タイマーを停止し実行時間を表示*/
  checkCudaErrors(hipEventRecord(stop, NULL));
  checkCudaErrors(hipEventSynchronize(stop));
  float msecTotal = 0.0f;
  checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));
  printf("Processing time: %f (msec)\n", msecTotal);
  
  /*再度タイマー開始*/
  checkCudaErrors(hipEventRecord(start, NULL));
  
  /*結果の領域確保とデバイス側からのメモリ転送*/
  host_result = (unsigned long int *)malloc(thread_size * sizeof(unsigned long int ));
  checkCudaErrors(hipMemcpy(host_result, device_result, thread_size * sizeof(unsigned long int ) , hipMemcpyDeviceToHost));
  
  /*タイマーを停止し実行時間を表示*/
  checkCudaErrors(hipEventRecord(stop, NULL));
  checkCudaErrors(hipEventSynchronize(stop));
  checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));
  printf("Memory copy time: %f (msec)\n", msecTotal);
  printf("Now Writing...\n");
  printf("%u\n",thread_size);
  for( unsigned long int  l = 0; l < thread_size; l++ ){ 
    fprintf(outputfile,"%llu\n", host_result[l]);
  }
  fclose(outputfile);
  
  /*ホスト・デバイスメモリの開放*/
  free(host_result);
  checkCudaErrors(hipFree(device_result));
  timer();
  
  /*終了処理*/
  hipDeviceReset();
  exit(0);
  
}

__global__ void thread_num(unsigned long int  *device_result){
  
  /*スレッドIDの割り当て*/
  /* blockDim = block size , threadIdx = 0~blockDim-1 , blockIdx = 0~grid size-1 , max thread = blockDim * max blockIdx + max threadIdx */
  unsigned long int  thread_idx = threadIdx.x+blockDim.x*blockIdx.x;
  unsigned long int  thread_idy = threadIdx.y+blockDim.y*blockIdx.y;
  unsigned long int  thread_idz = threadIdx.z+blockDim.z*blockIdx.z;
  unsigned long int thread_id = ( blockDim.x * (Grid_x - 1) + blockDim.x ) * ( blockDim.y * (Grid_y - 1) + blockDim.y ) * thread_idz + ( blockDim.x * (Grid_x - 1) + blockDim.x ) * thread_idy + thread_idx ;
  /*thread id*/
    device_result[thread_id] = thread_id;
}
