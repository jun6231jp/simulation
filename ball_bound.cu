#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <GL/gl.h>
#include <GL/glut.h>
#include <math.h>
#include <stdbool.h>
#include <omp.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include <hip/hip_runtime.h>

#define PI 3.141592653589793

#define cap 1000
#define ref 0.6
#define temp 4273
#define visc 1000000000
#define GRAV (6.674*0.00000000000000000001)
#define density (2.5 * 1000000000000)
#define M (4 / 3 * PI * reluc*reluc*reluc* density)

#define reluc 15
#define dev 12
#define nump 15
#define NUM_POINTS (nump * nump * nump)

#define X 0
#define Y 1
#define Z 2
#define INIT_WIDTH 500
#define INIT_HEIGHT 500
#define vision 20
#define Grid_x nump
#define Grid_y nump
#define Grid_z 1
#define Block_x nump
#define Block_y 1
#define Block_z 1

unsigned int num_points = (dev + 1) * (dev + 1);
unsigned int window_width = 150;
unsigned int window_height = 150;
double init_left = -10000;
double init_right = 10000;
double init_bottom = -10000;
double init_top = 10000;
double left, right, bottom, top;
float h_point[NUM_POINTS][3];
float v_point[NUM_POINTS][3];
float anim_time = 0.0f;
float anim_dt = 0.000000000001;
double phi = 30.0;
double theta = 30.0;
float light_pos[4];
int mouse_old_x, mouse_old_y;
bool motion_p;
double eye[3];
double center[3] = {0.0, 0.0, 0.0};
double up[3];
double ** point;
float (*d_point)[3];
float (*dv_point)[3];

__global__ void grav_v(float (*pos)[3], float(*vec)[3] , float time, float dt);
__global__ void grav_p(float (*pos)[3], float(*vec)[3] , float time, float dt);
//��{�֐��Q
double dot(double vec0[], double vec1[])
{
  return(vec0[X] * vec1[X] + vec0[Y] * vec1[Y] + vec0[Z] * vec1[Z]);
}
void cross(double vec0[], double vec1[], double vec2[])
{
  vec2[X] = vec0[Y] * vec1[Z] - vec0[Z] * vec1[Y];
  vec2[Y] = vec0[Z] * vec1[X] - vec0[X] * vec1[Z];
  vec2[Z] = vec0[X] * vec1[Y] - vec0[Y] * vec1[X];
}
void normVec(double vec[])
{
  double norm;
  norm = sqrt(vec[X] * vec[X] + vec[Y] * vec[Y] + vec[Z] * vec[Z]);
  vec[X] /= norm;
  vec[Y] /= norm;
  vec[Z] /= norm;
}
void normal(double p0[], double p1[], double p2[], double normal[])
{
  unsigned int i;
  double v0[3], v1[3];
  for (i = 0; i < 3; i++) {
    v0[i] = p2[i] - p1[i];
    v1[i] = p0[i] - p1[i];
  }
  cross(v0, v1, normal);
  normVec(normal);
}
int comp (const void *a, const void *b){
  return *(int *)a-*(int *)b;
}

//�d�͉e����̑��x������
__global__ void grav_v(float (*pos)[3],float(*vec)[3], float time, float dt)
{
  double xn,yn,zn,vx,vy,vz,dis,sq;
  unsigned int thread_idx = threadIdx.x+blockDim.x*blockIdx.x;
  unsigned int thread_idy = threadIdx.y+blockDim.y*blockIdx.y;
  unsigned int thread_idz = threadIdx.z+blockDim.z*blockIdx.z;
  unsigned int index = ( blockDim.x * (Grid_x - 1) + blockDim.x ) * ( blockDim.y * (Grid_y - 1) + blockDim.y ) * thread_idz + ( blockDim.x * (Grid_x - 1) + blockDim.x ) * thread_idy + thread_idx ;
  float v_buff[NUM_POINTS][3]={0};

  float alive[NUM_POINTS]={1};
  float r[NUM_POINTS]={reluc};
  float m[NUM_POINTS]={M};
  float s[NUM_POINTS]={visc};
  float e[NUM_POINTS]={ref};
  float T[NUM_POINTS]={temp};
  float J[NUM_POINTS]={cap*M*temp};

  float coltime[NUM_POINTS][2]={0};
  int colflag=0;
  xn = pos[index][0];
  yn = pos[index][1];
  zn = pos[index][2];
  vx = vec[index][0];
  vy = vec[index][1];
  vz = vec[index][2];
  v_buff[index][0]=vx;
  v_buff[index][1]=vy;
  v_buff[index][2]=vz;

  for (int i = 0 ; i < NUM_POINTS; i++)
    {
      sq = pow((double)(xn-pos[i][0]),2) + pow((double)(yn-pos[i][1]),2) + pow((double)(zn-pos[i][2]),2);
      dis = sqrt(sq);
      if (dis > r[index] + r[i])
	{
          //�^���G�l���M�[�Ɠ����G�l���M�[�̌��� 
	  J[i]-=0.5*m[index]*fabs((pow((double)vx,2)+pow((double)vy,2)+pow((double)vz,2))-(pow((double)(vx + ((pos[i][0]-xn)/dis)*(GRAV*m[index]*m[i]/sq)),2)+pow((double)(vy + ((pos[i][1]-yn)/dis)*(GRAV*m[index]*m[i]/sq)),2)+pow((double)(vz + ((pos[i][2]-zn)/dis)*(GRAV*m[index]*m[i]/sq)),2)));
	  //���x�X�V
	  vx = vx + ((pos[i][0]-xn)/dis)*(GRAV*m[index]*m[i]/sq);
	  vy = vy + ((pos[i][1]-yn)/dis)*(GRAV*m[index]*m[i]/sq);
	  vz = vz + ((pos[i][2]-zn)/dis)*(GRAV*m[index]*m[i]/sq);
	}
      else 
	{
	  if (i != index)
	    {
	      colflag=1;
	      //�Փ˂���܂ł̎��Ԃ��L�^�@TBD�@�@�������ɒ���
              coltime[i][1]=(float)i;
	      coltime[i][0]=(float)(sq/(pow((double)(vx-vec[index][0]),2)+pow((double)(vy-vec[index][1]),2)+pow((double)(vz-vec[index][2]),2)));
	    }
	}
    }
  if(colflag==1){
    //�Փ˂܂ł̎��Ԃ��C���f�b�N�X�\�[�g
    qsort(coltime, sizeof(coltime)/2/sizeof(float), 2*sizeof(float),comp);
    for (int i=NUM_POINTS-1 ; i>=0; i--){
      int colindex=(int)coltime[i][1];
      //�����W���͏������ق���D��
      float repul=e[index];
      if (e[colindex] < e[index]) {
	repul=e[colindex];
      }
      //���x�X�V
      v_buff[index][0]=((1+repul)*m[colindex]*vec[colindex][0]+(m[index]-repul*m[colindex])*v_buff[index][0])/(m[index]+m[colindex]);
      v_buff[index][1]=((1+repul)*m[colindex]*vec[colindex][1]+(m[index]-repul*m[colindex])*v_buff[index][1])/(m[index]+m[colindex]);
      v_buff[index][2]=((1+repul)*m[colindex]*vec[colindex][2]+(m[index]-repul*m[colindex])*v_buff[index][2])/(m[index]+m[colindex]);
      //�Փ˃G�l���M�[�̔M�G�l���M�[�ϊ�
      float Energy=0.5*(1-repul*repul)*(m[index]*())
      //�S���Ɣ����W���̍X�V

      //�����W�����������l�ȉ��Ȃ�z��
      
    }
  }
  __syncthreads();
  vec[index][0] = vx + v_buff[index][0];
  vec[index][1] = vy + v_buff[index][1];
  vec[index][2] = vz + v_buff[index][2];
  //�����G�l���M�[�Ɖ^���G�l���M�[����M�G�l���M�[�X�V                                                                
  T[index]=(J[index]-0.5*m[index]*(pow((double)vec[index][0],2)+pow((double)vec[index][1],2)+pow((double)vec[index][2],2)))/m[index]/caps;
  
}
//�d�͉e����̍��W������
__global__ void grav_p(float (*pos)[3], float(*vec)[3] , float time, float dt)
{
  double xn,yn,zn,vx,vy,vz;
  unsigned int thread_idx = threadIdx.x+blockDim.x*blockIdx.x;
  unsigned int thread_idy = threadIdx.y+blockDim.y*blockIdx.y;
  unsigned int thread_idz = threadIdx.z+blockDim.z*blockIdx.z;
  unsigned int index = ( blockDim.x * (Grid_x - 1) + blockDim.x ) * ( blockDim.y * (Grid_y - 1) + blockDim.y ) * thread_idz + ( blockDim.x * (Grid_x - 1) + blockDim.x ) * thread_idy + thread_idx ;
  xn = pos[index][0];
  yn = pos[index][1];
  zn = pos[index][2];
  vx = vec[index][0];
  vy = vec[index][1];
  vz = vec[index][2];
  pos[index][0] = xn + vx * dt;
  pos[index][1] = yn + vy * dt;
  pos[index][2] = zn + vz * dt;
}

// ���q�������ʒu�ɔz�u�D
void setInitialPosition(void)
{
  srand(12131);
  for (int i = 0; i < NUM_POINTS; i++) {
    if(rand()%2==1)
      {
	h_point[i][0] = (double)rand() / RAND_MAX * INIT_WIDTH / 20;
        h_point[i][1] = (double)rand() / RAND_MAX * INIT_WIDTH / 20;
        h_point[i][2] = (double)rand() / RAND_MAX * INIT_WIDTH / 20;
      }
    else
      {
	h_point[i][0] = -(double)rand() / RAND_MAX * INIT_WIDTH / 20;
        h_point[i][1] = -(double)rand() / RAND_MAX * INIT_WIDTH / 20;
        h_point[i][2] = -(double)rand() / RAND_MAX * INIT_WIDTH / 20;
      }
  }
  for (int i = 0; i < NUM_POINTS; i++) {
    v_point[i][0] = 0;
    v_point[i][1] = 0;
    v_point[i][2] = 0;
  }
  checkCudaErrors(hipMalloc((void**)&d_point, 3 * NUM_POINTS * sizeof(float)));
  checkCudaErrors(hipMalloc((void**)&dv_point, 3 * NUM_POINTS * sizeof(float)));
  checkCudaErrors(hipMemcpy(d_point, h_point, 3 * NUM_POINTS * sizeof(float) , hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(dv_point, v_point, 3 * NUM_POINTS * sizeof(float) , hipMemcpyHostToDevice));
}
//CUDA���s�֐�
void launchGPUKernel(unsigned int num_particles, float (*pos)[3], float (*vec)[3] , float time, float dt)
{
    dim3 grid(Grid_x,Grid_y,Grid_z);
    dim3 block(Block_x,Block_y,Block_z);
    grav_v<<<grid , block>>>(pos, vec, time, dt);
    grav_p<<<grid , block>>>(pos, vec, time, dt);
}
//�A�j���[�V��������
void runGPUKernel(void)
{
  launchGPUKernel(NUM_POINTS, d_point, dv_point , anim_time, anim_dt);
  checkCudaErrors(hipMemcpy(h_point, d_point, 3 * NUM_POINTS * sizeof(float) , hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(v_point, dv_point, 3 * NUM_POINTS * sizeof(float) , hipMemcpyDeviceToHost));
  anim_time += anim_dt;
}
//�r���[��`
void defineViewMatrix(double phi, double theta)
{
  unsigned int i;
  double c, s, xy_dist;
  double x_axis[3], y_axis[3], z_axis[3];

  // ���_�̐ݒ�D
  eye[Z] = sin(theta * PI / 180.0);
  xy_dist = cos(theta * PI / 180.0);
  c = cos(phi * PI / 180.0);
  s = sin(phi * PI / 180.0);
  eye[X] = xy_dist * c;
  eye[Y] = xy_dist * s;
  up[X] = - c * eye[Z];
  up[Y] = - s * eye[Z];
  up[Z] = s * eye[Y] + c * eye[X];
  normVec(up);
  
  // ���_�����_�Ƃ�����W�n�̒�`�D
  for (i = 0; i < 3; i++)
    {
      z_axis[i] = eye[i] - center[i];
    }
  normVec(z_axis);
  cross(up, z_axis, x_axis);
  normVec(x_axis);
  cross(z_axis, x_axis, y_axis);
  gluLookAt(eye[X], eye[Y], eye[Z], center[X], center[Y], center[Z], up[X], up[Y], up[Z]); 
}

void display(void)
{
    double nrml_vec[3];

    light_pos[0] = (float)eye[X];
    light_pos[1] = (float)eye[Y];
    light_pos[2] = (float)eye[Z];
    light_pos[3] = 0.0f;
    //CUDA�J�n
    runGPUKernel();

    // �����̐ݒ�
    glLightfv(GL_LIGHT0, GL_POSITION, light_pos);
    //glLightfv(GL_LIGHT0, GL_DIFFUSE, light_pos);
    glEnable(GL_LIGHTING);
    glMatrixMode(GL_PROJECTION);
    glFrustum(-1000000, 1000000, -1000000, 1000000, -1000000, 1000000); 
    glLoadIdentity();
    glOrtho(-vision, vision, -vision, vision, -1000, 1000);
    glViewport(0, 0, window_width, window_height);
    defineViewMatrix(phi, theta);
    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
    glBegin(GL_QUADS);
    
    //���̂��|���S���ō쐬
    for (int k = 0 ; k < NUM_POINTS ; k++)
      {
	for (int i = 0 ; i < dev + 1 ; i ++)
	  {
	    for (int j = 0 ; j < 2 * dev + 1 ; j++)
	      {
		normal(point[i * (dev-1) + j],point[(i + 1) * (dev-1) + j + 1],point[(i+1) * (dev-1) + j],nrml_vec);
		glNormal3dv(nrml_vec);
		glVertex3d(point[i * (dev-1) + j][X] + h_point[k][X], point[i * (dev-1) + j][Y] + h_point[k][Y], point[i * (dev-1) + j][Z] + h_point[k][Z]);
		glVertex3d(point[(i + 1) * (dev-1) + j][X] + h_point[k][X],point[(i + 1) * (dev-1) + j][Y] + h_point[k][Y],point[(i + 1) * (dev-1) + j][Z] + h_point[k][Z]);
		glVertex3d(point[(i + 1) * (dev-1) + j + 1][X] + h_point[k][X], point[(i + 1) * (dev-1) + j + 1][Y] + h_point[k][Y], point[(i + 1) * (dev-1) + j + 1][Z] + h_point[k][Z]);
		glVertex3d(point[i * (dev-1) + j + 1][X] + h_point[k][X],point[i * (dev-1) + j + 1][Y] + h_point[k][Y],point[i * (dev-1) + j + 1][Z] + h_point[k][Z]);
	      }
	  }
      }
    glEnd();
    glutSwapBuffers();
    glutPostRedisplay();
}

void mouse_button(int button, int state, int x, int y)
{
  if ((state == GLUT_DOWN) && (button == GLUT_LEFT_BUTTON))
    motion_p = true;
  else if (state == GLUT_UP)
    motion_p = false;
  mouse_old_x = x;
  mouse_old_y = y;
}
void mouse_motion(int x, int y)
{
  int dx, dy;
  dx = x - mouse_old_x;
  dy = y - mouse_old_y;
  if (motion_p) {
    phi -= dx * 0.2;
    theta += dy * 0.2;
  }
  mouse_old_x = x;
  mouse_old_y = y;
  glutPostRedisplay();
}
void resize (int width, int height)
{
  window_width = width;
  window_height = height;
}
bool initGL(void)
{
  glClearColor(0.0f, 0.0f , 0.0f, 0.5f);
  glEnable(GL_DEPTH_TEST);
  glClearDepth(1.0);
  glDepthFunc(GL_LESS);
  glEnable(GL_LIGHT0);
  return true;
}

int main(int argc, char** argv)
{
  double yangle,zangle;
  double r;

  point = (double **)malloc(sizeof(double *) * num_points);
  for (int i = 0 ; i < num_points ; i++)
    {
      point[i] = (double *)malloc(sizeof(double) * 3);
    } 
  for (int i = 0 ; i < dev + 1; i ++)
    {
      zangle = i * PI / dev;

      r=R * sin(zangle);
      for (int j = 0 ; j < dev + 1; j++)
	{
	  yangle=j * PI * 2 / dev;

	  point[i * dev + j][X] = r * sin(yangle);
	  point[i * dev + j][Y] = r * cos(yangle);
	  point[i * dev + j][Z] = R * cos(zangle);
	}
    }
    glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE | GLUT_DEPTH);
    glutInitWindowSize(INIT_WIDTH, INIT_HEIGHT);
    glutCreateWindow("3D CUDA Simulation");
    glutDisplayFunc(display);
    glutReshapeFunc(resize);
    glutMouseFunc(mouse_button);
    glutMotionFunc(mouse_motion);

    setInitialPosition();

    if (!initGL())
      return 1;
 
    glutMainLoop();
    hipFree(d_point);
    hipFree(dv_point);
    hipDeviceReset();
    for (int i = 0 ; i < num_points ; i++)
      {
	free (point[i]);
      }
    free (point);
    return 0;
}
