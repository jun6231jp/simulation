#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <GL/gl.h>
#include <GL/glut.h>
#include <math.h>
#include <stdbool.h>
#include <omp.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include <hip/hip_runtime.h>
#include <unistd.h>

#define PI 3.141592653589793

#define cap 1000
#define ref 0.9
#define temp 4273
#define visc 21
#define GRAV (6.674*0.00000000000000000001)
#define density (2.5 * 1000000000000)

#define rad 50
#define dev 12
#define M (4 / 3 * PI * rad*rad*rad* density)

#define X 0
#define Y 1
#define Z 2
#define ANIM 100000
#define scale 0.01
#define colmargin 1.05
#define R (rad * scale)
#define INIT_WIDTH 800
#define INIT_HEIGHT 800
#define vision 40
#define Grid_x 1 //block�Ԃ�syncthread�œ����ł��Ȃ�
#define Grid_y 1
#define Grid_z 1
#define Block_x 16
#define Block_y 4
#define Block_z 1

#define NUM_POINTS (Grid_x*Grid_y*Grid_z*Block_x*Block_y*Block_z)

unsigned int num_points = (dev + 1) * (dev + 1);
unsigned int window_width = 500;
unsigned int window_height = 500;
double init_left = -10000;
double init_right = 10000;
double init_bottom = -10000;
double init_top = 10000;
double left, right, bottom, top;
float h_point[NUM_POINTS][3];
float v_point[NUM_POINTS][3]={0};
float st_point[NUM_POINTS]={visc};
float e_point[NUM_POINTS]={ref};
float T_point[NUM_POINTS]={temp};
float J_point[NUM_POINTS]={cap*M*temp};
float anim_time = ANIM;
float anim_dt = 0.1;
double phi = 30.0;
double theta = 30.0;
float light_pos[4];
int mouse_old_x, mouse_old_y;
bool motion_p;
double eye[3];
double center[3] = {0.0, 0.0, 0.0};
double up[3];
double ** point;
float (*d_point)[3];
float (*dv_point)[3];
float (*dst_point);
float (*de_point);
float (*dT_point);
float (*dJ_point);

float colsynctime[NUM_POINTS][NUM_POINTS]={0.0};
float colsyncindex[NUM_POINTS][NUM_POINTS]={0.0};
float (*dcolsynctime)[NUM_POINTS];
float (*dcolsyncindex)[NUM_POINTS];

__global__ void grav_v(float (*pos)[3], float(*vec)[3] ,float(*sti),float(*e),float(*T),float(*J), float time, float dt,float(*coltime)[NUM_POINTS],float(*colindex)[NUM_POINTS]);
__global__ void grav_p(float (*pos)[3], float(*vec)[3] , float time, float dt);
/*
texture<float,2> timetocol;
texture<float,2> indextocol;
void bindTextures(float **coltime, float **colindex)
{
  hipChannelFormatDesc desc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
  hipBindTexture(0, timetocol, coltime, desc);
  hipBindTexture(0, indextocol,colindex, desc);
}
void unbindTextures(void)
{
  hipUnbindTexture(timetocol);
  hipUnbindTexture(indextocol);
}
*/

//��{�֐��Q
double dot(double vec0[], double vec1[])
{
  return(vec0[X] * vec1[X] + vec0[Y] * vec1[Y] + vec0[Z] * vec1[Z]);
}
void cross(double vec0[], double vec1[], double vec2[])
{
  vec2[X] = vec0[Y] * vec1[Z] - vec0[Z] * vec1[Y];
  vec2[Y] = vec0[Z] * vec1[X] - vec0[X] * vec1[Z];
  vec2[Z] = vec0[X] * vec1[Y] - vec0[Y] * vec1[X];
}
void normVec(double vec[])
{
  double norm;
  norm = sqrt(vec[X] * vec[X] + vec[Y] * vec[Y] + vec[Z] * vec[Z]);
  vec[X] /= norm;
  vec[Y] /= norm;
  vec[Z] /= norm;
}
void normal(double p0[], double p1[], double p2[], double normal[])
{
  unsigned int i;
  double v0[3], v1[3];
  for (i = 0; i < 3; i++) {
    v0[i] = p2[i] - p1[i];
    v1[i] = p0[i] - p1[i];
  }
  cross(v0, v1, normal);
  normVec(normal);
}

//�d�͉e����̑��x������
__global__ void grav_v(float (*pos)[3],float(*vec)[3],float(*sti),float(*e),float(*T),float(*J), float time, float dt,float(*coltime)[NUM_POINTS],float(*colindex)[NUM_POINTS])
{
  double xn,yn,zn,vx,vy,vz,dis,sq;
  unsigned int thread_idx = threadIdx.x+blockDim.x*blockIdx.x;
  unsigned int thread_idy = threadIdx.y+blockDim.y*blockIdx.y;
  unsigned int thread_idz = threadIdx.z+blockDim.z*blockIdx.z;
  unsigned int index = (blockDim.x * Grid_x) * (blockDim.y * Grid_y) * thread_idz + (blockDim.x * Grid_x) * thread_idy + thread_idx ;

  double v_buff[3]={0};
  //double coltime[NUM_POINTS][2]={0};
  int colnum=0;
  double gravity=0;
  xn = pos[index][0];
  yn = pos[index][1];
  zn = pos[index][2];
  vx = vec[index][0];
  vy = vec[index][1];
  vz = vec[index][2];
  v_buff[0]=vx;
  v_buff[1]=vy;
  v_buff[2]=vz;
  
  for (int i = 0 ; i < NUM_POINTS; i++)
    {
      sq = pow((double)(xn-pos[i][0]),2) + pow((double)(yn-pos[i][1]),2) + pow((double)(zn-pos[i][2]),2);
      gravity=GRAV*M/sq*scale*scale;
      dis = sqrt(sq);
      //�Փˈ�N������
      if (dis > 2 * R * colmargin  && i != index)
	{
	  //�����q�֗^����^���G�l���M�[�Ɠ����G�l���M�[�̌��� 
	  //J[index]-=0.5*M*fabs((float)(pow((double)vec[i][0]/scale,2)+pow((double)vec[i][1]/scale,2)+pow((double)vec[i][2]/scale,2))-(float)(pow((double)(vec[i][0]/scale + 0.5*((pos[i][0]-xn)/dis)*gravity*ANIM*ANIM),2)+pow((double)(vec[i][1]/scale + 0.5*((pos[i][1]-yn)/dis)*gravity*ANIM*ANIM),2)+pow((double)(vec[i][2]/scale + 0.5*((pos[i][2]-zn)/dis)*gravity*ANIM*ANIM),2)));
	  //���x�X�V
	  vx = vx + ((pos[i][0]-xn)/dis)*gravity*ANIM*scale;
	  vy = vy + ((pos[i][1]-yn)/dis)*gravity*ANIM*scale;
	  vz = vz + ((pos[i][2]-zn)/dis)*gravity*ANIM*scale;
	  colindex[index][i]=NUM_POINTS;
	}
      else if (i != index)
	{
	  //�Փˈ�N������̌o�߂̎��Ԃ��L�^�@TBD�@�@�������ɒ���
	  colindex[index][i]=i;
	  coltime[index][i]=(2*R*colmargin - dis)/(pow((double)(vx-vec[i][0]),2)+pow((double)(vy-vec[i][1]),2)+pow((double)(vz-vec[i][2]),2));
	  colnum++;
	}
      else
	{
          colindex[index][i]=NUM_POINTS;
	}
    }
  //__syncthreads();
  /*
  int syncchk=0;
  for (int k=0; k<7000; k++){
    for(int i=0;i<NUM_POINTS;i++){
      for(int j=0;j<NUM_POINTS;j++){
	if(colindex[i][j]==0){
	  syncchk=0;
	  break;
	}
	else{
	  syncchk=1;
	}
      }
      if(syncchk==0){
	break;
      }
    }
    if(syncchk==0){
      usleep(10);
    }
    else {
      break;
    }
  }
  */
  if(colnum>0)
    {
      //�Փˈ�N������̌o�ߎ��Ԃ��C���f�b�N�X�t���\�[�g
      double tmp[2]={0};
      for (int i = 0 ; i < NUM_POINTS; i++){
	for(int j = i+1; j < NUM_POINTS; j++){
	  if(coltime[index][i] > coltime[index][j]){
	    tmp[0]=coltime[index][i];
	    tmp[1]=colindex[index][i];
	    coltime[index][i]=coltime[index][j];
	    colindex[index][i]=colindex[index][j];
	    coltime[index][j]=tmp[0];
	    colindex[index][j]=tmp[1];
	  }
	}
      }
      //�Փˈ�N������̌o�ߎ��Ԃ��������̂��珈��
      for (int i=NUM_POINTS-1 ; i>=NUM_POINTS-colnum; i--){
	int coldex=coltime[i][1];
	float repul=0;
	if (coldex != index) {
	  repul=e[index];
	  //�����W���͏������ق���D��     
	  if (e[coldex] < e[index]) {
	    repul=e[coldex];
	  }
	  //���x�X�V
	  v_buff[0]=(double)((1+repul)*M*vec[coldex][0]+(M-repul*M)*v_buff[0])/(M+M);
	  v_buff[1]=(double)((1+repul)*M*vec[coldex][1]+(M-repul*M)*v_buff[1])/(M+M);
	  v_buff[2]=(double)((1+repul)*M*vec[coldex][2]+(M-repul*M)*v_buff[2])/(M+M);
	  //�Փ˃G�l���M�[��m^2/3*sti�̔�ŕ��z���M�G�l���M�[�ϊ�
	  //double Energy=0.5*(1-repul*repul)*(M*(pow((double)vx/scale,2)+pow((double)vy/scale,2)+pow((double)vz/scale,2)) + M*(pow((double)vec[coldex][0]/scale,2)+pow((double)vec[coldex][1]/scale,2)+pow((double)vec[coldex][2]/scale,2)));
	  //J[index]+=Energy * pow((double)M,0.667) * pow(10.0,(double)sti[index]) / (pow((double)M,0.667) * pow(10.0,(double)sti[index]) + pow((double)M,0.667) * pow(10.0,(double)sti[coldex]));
	  //T[index]=(J[index]-0.5*M*(pow((double)v_buff[0],2)+pow((double)v_buff[1],2)+pow((double)v_buff[2],2)))/M/cap;	
	  //�S���Ɣ����W���̍X�V
	  //e[index] = e[index] * (visc+(temp/100)-(T[index]/100)-0.5*log(M))/(visc-0.5*log(M));
	  //sti[index] = visc - (T[index] - temp / 100);
	}
      }
    }
  for(int i=0;i<NUM_POINTS;i++){
    coltime[index][i]=0;
    colindex[index][i]=0;
  }
  //  __syncthreads();
  /*
  for (int k=0; k<7000; k++){
    for(int i=0;i<NUM_POINTS;i++){
      for(int j=0;j<NUM_POINTS;j++){
        if(colindex[i][j]!=0){
          syncchk=1;
          break;
        }
        else{
          syncchk=0;
        }
      }
      if(syncchk==1){
        break;
      }
    }
    if(syncchk==1){
      usleep(10);
    }
    else {
      break;
    }
  }
  */
  if (colnum>0)
    {
      vec[index][0] = (float)v_buff[0];
      vec[index][1] = (float)v_buff[1];
      vec[index][2] = (float)v_buff[2];
    }
  else
    {
      vec[index][0] = (float)vx;
      vec[index][1] = (float)vy;
      vec[index][2] = (float)vz;
    }
  //�����G�l���M�[�Ɖ^���G�l���M�[����M�G�l���M�[�X�V                                                                
  //T[index]=(J[index]-0.5*M*(pow((double)vec[index][0]/scale,2)+pow((double)vec[index][1]/scale,2)+pow((double)vec[index][2]/scale,2)))/M/cap;
  //�S���Ɣ����W���̍X�V 
  //e[index] = e[index] * (visc+(temp/100)-(T[index]/100)-0.5*log(M))/(visc-0.5*log(M));
  //sti[index] = visc - (T[index] - temp / 100);
}

//�d�͉e����̍��W������
__global__ void grav_p(float (*pos)[3], float(*vec)[3] , float time, float dt)
{
  double xn,yn,zn,vx,vy,vz;
  unsigned int thread_idx = threadIdx.x+blockDim.x*blockIdx.x;
  unsigned int thread_idy = threadIdx.y+blockDim.y*blockIdx.y;
  unsigned int thread_idz = threadIdx.z+blockDim.z*blockIdx.z;
  unsigned int index = ( blockDim.x * (Grid_x - 1) + blockDim.x ) * ( blockDim.y * (Grid_y - 1) + blockDim.y ) * thread_idz + ( blockDim.x * (Grid_x - 1) + blockDim.x ) * thread_idy + thread_idx ;
  xn = pos[index][0];
  yn = pos[index][1];
  zn = pos[index][2];
  vx = vec[index][0];
  vy = vec[index][1];
  vz = vec[index][2];
  pos[index][0] = xn + vx * dt;
  pos[index][1] = yn + vy * dt;
  pos[index][2] = zn + vz * dt;
}

// ���q�������ʒu�ɔz�u�D
void setInitialPosition(void)
{
  srand(12131);
  for (int i = 0; i < NUM_POINTS; i++) {
    if(rand()%2==1)
      {
	h_point[i][0] = (double)rand() / RAND_MAX * INIT_WIDTH / 10;
        h_point[i][1] = (double)rand() / RAND_MAX * INIT_WIDTH / 10;
        h_point[i][2] = (double)rand() / RAND_MAX * INIT_WIDTH / 10;
      }
    else
      {
	h_point[i][0] = -(double)rand() / RAND_MAX * INIT_WIDTH / 10;
        h_point[i][1] = -(double)rand() / RAND_MAX * INIT_WIDTH / 10;
        h_point[i][2] = -(double)rand() / RAND_MAX * INIT_WIDTH / 10;
      }
  }
  
  checkCudaErrors(hipMalloc((void**)&d_point, 3 * NUM_POINTS * sizeof(float)));
  checkCudaErrors(hipMalloc((void**)&dv_point, 3 * NUM_POINTS * sizeof(float)));
  checkCudaErrors(hipMalloc((void**)&dst_point, NUM_POINTS * sizeof(float)));
  checkCudaErrors(hipMalloc((void**)&de_point, NUM_POINTS * sizeof(float)));
  checkCudaErrors(hipMalloc((void**)&dT_point, NUM_POINTS * sizeof(float)));
  checkCudaErrors(hipMalloc((void**)&dJ_point, NUM_POINTS * sizeof(float)));
  checkCudaErrors(hipMalloc((void**)&dcolsynctime, NUM_POINTS*NUM_POINTS * sizeof(float)));
  checkCudaErrors(hipMalloc((void**)&dcolsyncindex, NUM_POINTS*NUM_POINTS * sizeof(float)));
  checkCudaErrors(hipMemcpy(d_point, h_point, 3 * NUM_POINTS * sizeof(float) , hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(dv_point, v_point, 3 * NUM_POINTS * sizeof(float) , hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(dst_point, st_point, NUM_POINTS * sizeof(float) , hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(de_point, e_point, NUM_POINTS * sizeof(float) , hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(dT_point, T_point, NUM_POINTS * sizeof(float) , hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(dJ_point, J_point, NUM_POINTS * sizeof(float) , hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(dcolsynctime, colsynctime, NUM_POINTS*NUM_POINTS * sizeof(float) , hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(dcolsyncindex, colsyncindex, NUM_POINTS*NUM_POINTS * sizeof(float) , hipMemcpyHostToDevice));
}
//CUDA���s�֐�
void launchGPUKernel(unsigned int num_particles, float (*pos)[3], float (*vec)[3] ,float(*sti),float(*e),float(*T),float(*J), float time, float dt,float (*synctime)[NUM_POINTS],float (*syncindex)[NUM_POINTS])
{
    dim3 grid(Grid_x,Grid_y,Grid_z);
    dim3 block(Block_x,Block_y,Block_z);
    grav_v<<<grid , block>>>(pos, vec, sti, e, T, J, time, dt, synctime, syncindex);
    grav_p<<<grid , block>>>(pos, vec, time, dt);
}
//�A�j���[�V��������
void runGPUKernel(void)
{
  launchGPUKernel(NUM_POINTS, d_point, dv_point ,dst_point, de_point,dT_point,dJ_point, anim_time, anim_dt,dcolsynctime,dcolsyncindex);
  checkCudaErrors(hipMemcpy(h_point, d_point, 3 * NUM_POINTS * sizeof(float) , hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(v_point, dv_point, 3 * NUM_POINTS * sizeof(float) , hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(st_point, dst_point, NUM_POINTS * sizeof(float) , hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(e_point, de_point, NUM_POINTS * sizeof(float) , hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(T_point, dT_point, NUM_POINTS * sizeof(float) , hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(J_point, dJ_point, NUM_POINTS * sizeof(float) , hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(colsynctime,dcolsynctime, NUM_POINTS*NUM_POINTS * sizeof(float) , hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(colsyncindex,dcolsyncindex, NUM_POINTS*NUM_POINTS * sizeof(float) , hipMemcpyDeviceToHost));

  anim_time += anim_dt;
}
//�r���[��`
void defineViewMatrix(double phi, double theta)
{
  unsigned int i;
  double c, s, xy_dist;
  double x_axis[3], y_axis[3], z_axis[3];

  // ���_�̐ݒ�D
  eye[Z] = sin(theta * PI / 180.0);
  xy_dist = cos(theta * PI / 180.0);
  c = cos(phi * PI / 180.0);
  s = sin(phi * PI / 180.0);
  eye[X] = xy_dist * c;
  eye[Y] = xy_dist * s;
  up[X] = - c * eye[Z];
  up[Y] = - s * eye[Z];
  up[Z] = s * eye[Y] + c * eye[X];
  normVec(up);
  
  // ���_�����_�Ƃ�����W�n�̒�`�D
  for (i = 0; i < 3; i++)
    {
      z_axis[i] = eye[i] - center[i];
    }
  normVec(z_axis);
  cross(up, z_axis, x_axis);
  normVec(x_axis);
  cross(z_axis, x_axis, y_axis);
  gluLookAt(eye[X], eye[Y], eye[Z], center[X], center[Y], center[Z], up[X], up[Y], up[Z]); 
}

void display(void)
{
    double nrml_vec[3];

    light_pos[0] = (float)eye[X];
    light_pos[1] = (float)eye[Y];
    light_pos[2] = (float)eye[Z];
    light_pos[3] = 0.0f;
    //CUDA�J�n
    runGPUKernel();

    // �����̐ݒ�
    glLightfv(GL_LIGHT0, GL_POSITION, light_pos);
    //glLightfv(GL_LIGHT0, GL_DIFFUSE, light_pos);
    glEnable(GL_LIGHTING);
    glMatrixMode(GL_PROJECTION);
    glFrustum(-1000000, 1000000, -1000000, 1000000, -1000000, 1000000); 
    glLoadIdentity();
    glOrtho(-vision, vision, -vision, vision, -1000, 1000);
    glViewport(0, 0, window_width, window_height);
    defineViewMatrix(phi, theta);
    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
    glBegin(GL_QUADS);
    
    //���̂��|���S���ō쐬 TBD ���^�{�[��
    for (int k = 0 ; k < NUM_POINTS ; k++)
      {
	for (int i = 0 ; i < dev + 1 ; i ++)
	  {
	    for (int j = 0 ; j < 2 * dev + 1 ; j++)
	      {
		normal(point[i * (dev-1) + j],point[(i + 1) * (dev-1) + j + 1],point[(i+1) * (dev-1) + j],nrml_vec);
		glNormal3dv(nrml_vec);
		glVertex3d(point[i * (dev-1) + j][X] + h_point[k][X], point[i * (dev-1) + j][Y] + h_point[k][Y], point[i * (dev-1) + j][Z] + h_point[k][Z]);
		glVertex3d(point[(i + 1) * (dev-1) + j][X] + h_point[k][X],point[(i + 1) * (dev-1) + j][Y] + h_point[k][Y],point[(i + 1) * (dev-1) + j][Z] + h_point[k][Z]);
		glVertex3d(point[(i + 1) * (dev-1) + j + 1][X] + h_point[k][X], point[(i + 1) * (dev-1) + j + 1][Y] + h_point[k][Y], point[(i + 1) * (dev-1) + j + 1][Z] + h_point[k][Z]);
		glVertex3d(point[i * (dev-1) + j + 1][X] + h_point[k][X],point[i * (dev-1) + j + 1][Y] + h_point[k][Y],point[i * (dev-1) + j + 1][Z] + h_point[k][Z]);
	      }
	  }
      }
    glEnd();
    glutSwapBuffers();
    glutPostRedisplay();
}

void mouse_button(int button, int state, int x, int y)
{
  if ((state == GLUT_DOWN) && (button == GLUT_LEFT_BUTTON))
    motion_p = true;
  else if (state == GLUT_UP)
    motion_p = false;
  mouse_old_x = x;
  mouse_old_y = y;
}
void mouse_motion(int x, int y)
{
  int dx, dy;
  dx = x - mouse_old_x;
  dy = y - mouse_old_y;
  if (motion_p) {
    phi -= dx * 0.2;
    theta += dy * 0.2;
  }
  mouse_old_x = x;
  mouse_old_y = y;
  glutPostRedisplay();
}
void resize (int width, int height)
{
  window_width = width;
  window_height = height;
}
bool initGL(void)
{
  glClearColor(0.0f, 0.0f , 0.0f, 0.5f);
  glEnable(GL_DEPTH_TEST);
  glClearDepth(1.0);
  glDepthFunc(GL_LESS);
  glEnable(GL_LIGHT0);
  return true;
}

int main(int argc, char** argv)
{
  double yangle,zangle;
  double r;

  point = (double **)malloc(sizeof(double *) * num_points);
  for (int i = 0 ; i < num_points ; i++)
    {
      point[i] = (double *)malloc(sizeof(double) * 3);
    } 
  for (int i = 0 ; i < dev + 1; i ++)
    {
      zangle = i * PI / dev;

      r=R * sin(zangle);
      for (int j = 0 ; j < dev + 1; j++)
	{
	  yangle=j * PI * 2 / dev;

	  point[i * dev + j][X] = r * sin(yangle);
	  point[i * dev + j][Y] = r * cos(yangle);
	  point[i * dev + j][Z] = R * cos(zangle);
	}
    }
    glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE | GLUT_DEPTH);
    glutInitWindowSize(INIT_WIDTH, INIT_HEIGHT);
    glutCreateWindow("3D CUDA Simulation");
    glutDisplayFunc(display);
    glutReshapeFunc(resize);
    glutMouseFunc(mouse_button);
    glutMotionFunc(mouse_motion);

    setInitialPosition();

    if (!initGL())
      return 1;
 
    glutMainLoop();
    hipFree(d_point);
    hipFree(dv_point);
    hipFree(dst_point);
    hipFree(de_point);
    hipFree(dT_point);
    hipFree(dJ_point);

    hipDeviceReset();
    for (int i = 0 ; i < num_points ; i++)
      {
	free (point[i]);
	hipFree(dcolsynctime[i]);
	hipFree(dcolsyncindex[i]);
      }
    free (point);
    hipFree(dcolsynctime);
    hipFree(dcolsyncindex);

    return 0;
}
