#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <GL/glew.h>
#include <GL/gl.h>
#include <GL/glut.h>
#include <GL/freeglut.h>
#include <math.h>
#include <stdbool.h>
#include <omp.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>


#define PI 3.141592653589793
#define cap 1000
#define ref 0.9
#define temp 3000
#define visc 9
#define GRAV (6.674*0.00000000000000000001)
#define density (2.5 * 1000000000000)
#define cool (sigma*4*PI*rad*rad*1000000)
#define MOONOFFSET_X (INIT_WIDTH/vision/2)
#define MOONOFFSET_Y (INIT_WIDTH/vision*2)
#define MOONOFFSET_Z (INIT_HEIGHT/vision)
#define dev 360
#define resol 100
#define hollow 20
#define X 0
#define Y 1
#define Z 2
#define ANIM_START 0
#define ANIM 20
#define scale 0.01
#define colmargin 1.05
#define R (rad * scale)
#define INIT_WIDTH 800
#define INIT_HEIGHT 800
#define vision 40
#define Grid_y 2
#define Grid_z 1
#define Block_x 2
#define Block_y 1
#define Block_z 1

#define NUM_POINTS (Grid_x*Grid_y*Grid_z*Block_x*Block_y*Block_z)

unsigned int dev_points = dev + 1;
unsigned int window_width = INIT_WIDTH;
unsigned int window_height = INIT_HEIGHT;
double vision_size = vision;
float right_motion=0;
float up_motion=0;
double left, right, bottom, top;
float h_point[NUM_POINTS][3];
float v_point[NUM_POINTS][3];
float st_point[NUM_POINTS];
float e_point[NUM_POINTS];
float J_point[NUM_POINTS];
float h_buff[NUM_POINTS][3]={0};
float anim_time = ANIM_START;
float anim_dt = ANIM;
double phi = 30.0;
double theta = 30.0;
float light_pos[4];
int mouse_old_x, mouse_old_y;
bool motion_p;
bool motion_w;
double eye[3];
double center[3] = {0.0, 0.0, 0.0};
double up[3];
double ** point;
float (*d_point)[3];
float (*dv_point)[3];
float (*dst_point);
float (*de_point);
float (*dJ_point);
float (*v_buff)[3];
float colsynctime[NUM_POINTS][NUM_POINTS]={0};
int colsyncindex[NUM_POINTS][NUM_POINTS]={0};
float (*dcolsynctime)[NUM_POINTS];
int (*dcolsyncindex)[NUM_POINTS];
__global__ void grav_coldetect(float(*pos)[3],float(*vec)[3],float(*coltime)[NUM_POINTS],int(*colindex)[NUM_POINTS]);
__global__ void grav_colv(float(*pos)[3],float(*vec)[3],float(*v_buff)[3],float(*sti),float(*e),float(*J),float(*coltime)[NUM_POINTS],int(*colindex)[NUM_POINTS]);
__global__ void grav_v(float(*pos)[3],float(*vec)[3],float(*v_buff)[3],int(*colindex)[NUM_POINTS]);
__global__ void grav_vupdate(float(*vec)[3],float(*v_buff)[3]);
__global__ void buff_clear(float(*v_buff)[3],float(*coltime)[NUM_POINTS],int(*colindex)[NUM_POINTS]);
__global__ void grav_p(float (*pos)[3], float(*vec)[3]);

double dot(double vec0[], double vec1[])
{
  return(vec0[X] * vec1[X] + vec0[Y] * vec1[Y] + vec0[Z] * vec1[Z]);
}
void cross(double vec0[], double vec1[], double vec2[])
{
  vec2[X] = vec0[Y] * vec1[Z] - vec0[Z] * vec1[Y];
  vec2[Y] = vec0[Z] * vec1[X] - vec0[X] * vec1[Z];
  vec2[Z] = vec0[X] * vec1[Y] - vec0[Y] * vec1[X];
}
void normVec(double vec[])
{
  double norm;
  norm = sqrt(vec[X] * vec[X] + vec[Y] * vec[Y] + vec[Z] * vec[Z]);
  vec[X] /= norm;
  vec[Y] /= norm;
  vec[Z] /= norm;
}
void normal(double p0[], double p1[], double p2[], double normal[])
{
  unsigned int i;
  double v0[3], v1[3];
  for (i = 0; i < 3; i++) {
    v0[i] = p2[i] - p1[i];
    v1[i] = p0[i] - p1[i];
  }
  cross(v0, v1, normal);
  normVec(normal);
}
//衝突検知
__global__ void grav_coldetect(float(*pos)[3],float(*vec)[3],float(*coltime)[NUM_POINTS],int(*colindex)[NUM_POINTS])
{
  float xn,yn,zn,vx,vy,vz,dis,sq;
  unsigned int thread_idx = threadIdx.x+blockDim.x*blockIdx.x;
  unsigned int thread_idy = threadIdx.y+blockDim.y*blockIdx.y;
  unsigned int thread_idz = threadIdx.z+blockDim.z*blockIdx.z;
  unsigned int index = (blockDim.x * Grid_x) * (blockDim.y * Grid_y) * thread_idz + (blockDim.x * Grid_x) * thread_idy + thread_idx ;
  float rvec[3]={0};
  xn = pos[index][0];
  yn = pos[index][1];
  zn = pos[index][2];
  vx = vec[index][0];
  vy = vec[index][1];
  vz = vec[index][2];
  for (int i = 0 ; i < NUM_POINTS; i++)
    {
      sq = (float)pow((double)(xn-pos[i][0]),2) + pow((double)(yn-pos[i][1]),2) + pow((double)(zn-pos[i][2]),2);
      dis = (float)sqrt((double)sq);
      rvec[0]=(pos[i][0]-xn)/dis;
      rvec[1]=(pos[i][1]-yn)/dis;
      rvec[2]=(pos[i][2]-zn)/dis;
      if (dis > 2 * R * colmargin  && i != index)
        {
          colindex[index][i]=NUM_POINTS;
        }
      else if (dis <= 2 * R * colmargin && i != index)
        {
          colindex[index][i]=i;
          coltime[index][i]=(2*R*colmargin-dis)/((vx-vec[i][0])*rvec[0]+(vy-vec[i][1])*rvec[1]+(vz-vec[i][2])*rvec[2]);
        }
      else
        {
          colindex[index][i]=NUM_POINTS;
        }
    }
}
//衝突後の速度ベクトル計算
__global__ void grav_colv(float(*pos)[3],float(*vec)[3],float(*v_buff)[3],float(*sti),float(*e),float(*J),float(*coltime)[NUM_POINTS],int(*colindex)[NUM_POINTS])
{
  float xn,yn,zn,sq,dis;
  unsigned int thread_idx = threadIdx.x+blockDim.x*blockIdx.x;
  unsigned int thread_idy = threadIdx.y+blockDim.y*blockIdx.y;
  unsigned int thread_idz = threadIdx.z+blockDim.z*blockIdx.z;
  unsigned int index = (blockDim.x * Grid_x) * (blockDim.y * Grid_y) * thread_idz + (blockDim.x * Grid_x) * thread_idy + thread_idx ;
  int colnum=0;
  float tmptime=0;
  int tmpindex=0;
  int coldex=0;
  float repul=0;
  float rvec[3]={0};
  float Vl[3]={0};
  float Vr[3]={0};
  float Vh[3]={0};
  float vl_buff[3]={0};
  float vr_buff[3]={0};
  float vcol_buff[3]={0};
  float dotV=0;
  xn = pos[index][0];
  yn = pos[index][1];
  zn = pos[index][2];
  vl_buff[0]=vec[index][0];
  vl_buff[1]=vec[index][1];
  vl_buff[2]=vec[index][2];
  for (int i = 0 ; i < NUM_POINTS; i++){
    if(colindex[index][i]!=NUM_POINTS){
      colnum++;
    }
  }
  if(colnum>0){
      for (int i = 0 ; i < NUM_POINTS; i++){
        for(int j = i+1; j < NUM_POINTS; j++){
          if(coltime[index][i] > coltime[index][j]){
            tmptime=coltime[index][i];
            tmpindex=colindex[index][i];
            coltime[index][i]=coltime[index][j];
            colindex[index][i]=colindex[index][j];
            coltime[index][j]=tmptime;
            colindex[index][j]=tmpindex;
          }
        }
      }
      for (int i=NUM_POINTS-1 ; i>=NUM_POINTS-colnum; i--){
        coldex=colindex[index][i];
        sq = (float)pow((double)(xn-pos[coldex][0]),2)+pow((double)(yn-pos[coldex][1]),2)+pow((double)(zn-pos[coldex][2]),2);
        dis = (float)sqrt((double)sq);
        rvec[0]=(pos[coldex][0]-xn)/dis;
        rvec[1]=(pos[coldex][1]-yn)/dis;
        rvec[2]=(pos[coldex][2]-zn)/dis;
        dotV=rvec[0]*vl_buff[0]+rvec[1]*vl_buff[1]+rvec[2]*vl_buff[2];
        Vl[0]=dotV*rvec[0];
        Vl[1]=dotV*rvec[1];
        Vl[2]=dotV*rvec[2];
        dotV=rvec[0]*vec[coldex][0]+rvec[1]*vec[coldex][1]+rvec[2]*vec[coldex][2];
        Vr[0]=dotV*rvec[0];
        Vr[1]=dotV*rvec[1];
        Vr[2]=dotV*rvec[2];
        Vh[0]=vl_buff[0]-Vl[0];
        Vh[1]=vl_buff[1]-Vl[1];
        Vh[2]=vl_buff[2]-Vl[2];
        repul=e[index];
        if (e[coldex] < e[index]) {
          repul=e[coldex];
        }
        vcol_buff[0]=Vh[0]+((1+repul)*Vr[0]+(1-repul)*Vl[0])/2;
        vcol_buff[1]=Vh[1]+((1+repul)*Vr[1]+(1-repul)*Vl[1])/2;
        vcol_buff[2]=Vh[2]+((1+repul)*Vr[2]+(1-repul)*Vl[2])/2;
        vr_buff[0]=vec[coldex][0]-Vr[0]+((1+repul)*Vl[0]+(1-repul)*Vr[0])/2;
        vr_buff[1]=vec[coldex][1]-Vr[1]+((1+repul)*Vl[1]+(1-repul)*Vr[1])/2;
        vr_buff[2]=vec[coldex][2]-Vr[2]+((1+repul)*Vl[2]+(1-repul)*Vr[2])/2;
        double Energy=0.5*M*(pow(vec[coldex][0],2)+pow(vec[coldex][1],2)+pow(vec[coldex][2],2)+pow(vl_buff[0],2)+pow(vl_buff[1],2)+pow(vl_buff[2],2) - (pow(vcol_buff[0],2)+pow(vcol_buff[1],2)+pow(vcol_buff[2],2)+pow(vr_buff[0],2)+pow(vr_buff[1],2)+pow(vr_buff[2],2))) / pow(scale,2) * 1000000;
        J[index] += Energy / (pow(10.0,(double)(sti[index]-sti[coldex]))+1);
        if (J[index] > M * cap * 10000000){
          J[index] = M * cap * 10000000;
        }
        vl_buff[0]=vcol_buff[0];
        vl_buff[1]=vcol_buff[1];
        vl_buff[2]=vcol_buff[2];
        e[index] = 1 - ((1-ref)/temp * J[index]/M/cap);
        if ( e[index] < 0 ){ e[index] = 0; }
        else{ e[index] = 1; }
        sti[index] = visc - ((J[index]/M/cap - temp) / 100);
      }
      v_buff[index][0]=vl_buff[0];
      v_buff[index][1]=vl_buff[1];
      v_buff[index][2]=vl_buff[2];
  }
  J[index]-=cool*(J[index]/M/cap)*(J[index]/M/cap)*(J[index]/M/cap)*(J[index]/M/cap)*ANIM;
  if (J[index] < 0) {
    J[index] = 0;
  }
}
//重力影響後の速度ベクトル計算
__global__ void grav_v(float(*pos)[3],float(*vec)[3],float(*v_buff)[3],int(*colindex)[NUM_POINTS])
{
  float xn,yn,zn,vx,vy,vz,sq,dis;
  unsigned int thread_idx = threadIdx.x+blockDim.x*blockIdx.x;
  unsigned int thread_idy = threadIdx.y+blockDim.y*blockIdx.y;
  unsigned int thread_idz = threadIdx.z+blockDim.z*blockIdx.z;
  unsigned int index = (blockDim.x * Grid_x) * (blockDim.y * Grid_y) * thread_idz + (blockDim.x * Grid_x) * thread_idy + thread_idx ;
  int colnum=0;
  float gravity=0;
  xn = pos[index][0];
  yn = pos[index][1];
  zn = pos[index][2];
  for (int i = 0 ; i < NUM_POINTS; i++){
    if(colindex[index][i]!=NUM_POINTS){
      colnum++;
    }
  }
  if(colnum==0){
    vx = vec[index][0];
    vy = vec[index][1];
    vz = vec[index][2];
    for (int i = 0 ; i < NUM_POINTS; i++){
      if (i!=index) {
        sq = (float)pow((double)(xn-pos[i][0]),2) + pow((double)(yn-pos[i][1]),2) + pow((double)(zn-pos[i][2]),2);
        gravity=GRAV*M/sq*scale*scale;
        dis = (float)sqrt((double)sq);
        vx += ((pos[i][0]-xn)/dis)*gravity*ANIM*scale;
        vy += ((pos[i][1]-yn)/dis)*gravity*ANIM*scale;
        vz += ((pos[i][2]-zn)/dis)*gravity*ANIM*scale;
      }
    }
  }
  else {
    vx = v_buff[index][0];
    vy = v_buff[index][1];
    vz = v_buff[index][2];

    for (int i = 0 ; i < NUM_POINTS; i++){
      sq = (float)pow((double)(xn-pos[i][0]),2) + pow((double)(yn-pos[i][1]),2) + pow((double)(zn-pos[i][2]),2);
      gravity=GRAV*M/sq*scale*scale;
      dis = (float)sqrt((double)sq);
      if(dis > 2 * R * colmargin) {
        vx += ((pos[i][0]-xn)/dis)*gravity*ANIM*scale;
        vy += ((pos[i][1]-yn)/dis)*gravity*ANIM*scale;
        vz += ((pos[i][2]-zn)/dis)*gravity*ANIM*scale;
      }
    }

  }
  v_buff[index][0] = vx;
  v_buff[index][1] = vy;
  v_buff[index][2] = vz;
}
//速度ベクトル更新
__global__ void grav_vupdate(float(*vec)[3],float(*v_buff)[3])
{
  unsigned int thread_idx = threadIdx.x+blockDim.x*blockIdx.x;
  unsigned int thread_idy = threadIdx.y+blockDim.y*blockIdx.y;
  unsigned int thread_idz = threadIdx.z+blockDim.z*blockIdx.z;
  unsigned int index = (blockDim.x * Grid_x) * (blockDim.y * Grid_y) * thread_idz + (blockDim.x * Grid_x) * thread_idy + thread_idx ;
  vec[index][0]=v_buff[index][0];
  vec[index][1]=v_buff[index][1];
  vec[index][2]=v_buff[index][2];
}
//衝突検知用バッファクリア
__global__ void buff_clear(float(*v_buff)[3],float(*coltime)[NUM_POINTS],int(*colindex)[NUM_POINTS])
{
  unsigned int thread_idx = threadIdx.x+blockDim.x*blockIdx.x;
  unsigned int thread_idy = threadIdx.y+blockDim.y*blockIdx.y;
  unsigned int thread_idz = threadIdx.z+blockDim.z*blockIdx.z;
  unsigned int index = (blockDim.x * Grid_x) * (blockDim.y * Grid_y) * thread_idz + (blockDim.x * Grid_x) * thread_idy + thread_idx ;
  for (int i=0; i < 3; i++){
    v_buff[index][i]=0;
  }
  for (int i=0; i < NUM_POINTS; i++){
    coltime[index][i]=0;
    colindex[index][i]=NUM_POINTS;
  }
}
//重力影響後の位置更新
__global__ void grav_p(float(*pos)[3], float(*vec)[3])
{
  float xn,yn,zn,vx,vy,vz;
  unsigned int thread_idx = threadIdx.x+blockDim.x*blockIdx.x;
  unsigned int thread_idy = threadIdx.y+blockDim.y*blockIdx.y;
  unsigned int thread_idz = threadIdx.z+blockDim.z*blockIdx.z;
  unsigned int index = ( blockDim.x * (Grid_x - 1) + blockDim.x ) * ( blockDim.y * (Grid_y - 1) + blockDim.y ) * thread_idz + ( blockDim.x * (Grid_x - 1) + blockDim.x ) * thread_idy + thread_idx ;
  xn = pos[index][0];
  yn = pos[index][1];
  zn = pos[index][2];
  vx = vec[index][0];
  vy = vec[index][1];
  vz = vec[index][2];
  pos[index][0] = xn + vx * ANIM;
  pos[index][1] = yn + vy * ANIM;
  pos[index][2] = zn + vz * ANIM;
}

void setInitialPosition(void)
{
  int earth_points = NUM_POINTS - (NUM_POINTS/16);
  for (int i = 0; i < NUM_POINTS; i++) {
      for (int j = 0 ; j < 3 ; j++){
        h_point[i][j] = (float)(rand()-rand()) / RAND_MAX * INIT_WIDTH/vision*2 ;
        v_point[i][j] = 0;
        h_buff[i][j] = 0;
      }

    /*地球と隕石を分離して配置
    if(i < earth_points){
      for (int j = 0 ; j < 3 ; j++){
        h_point[i][j] = (float)(rand()-rand()) / RAND_MAX * INIT_WIDTH/vision/2 ;
        v_point[i][j] = 0;
        h_buff[i][j] = 0;
      }
    }
    else {
      h_point[i][0] = (float)(rand()-rand()) / RAND_MAX * INIT_WIDTH/vision/4 + MOONOFFSET_X;
      h_point[i][1] = (float)(rand()-rand()) / RAND_MAX * INIT_WIDTH/vision/4 + MOONOFFSET_Y;
      h_point[i][2] = (float)(rand()-rand()) / RAND_MAX * INIT_WIDTH/vision/4 + MOONOFFSET_Z;
      v_point[i][0] = -(MOONOFFSET_X*scale/ANIM)/4;
      v_point[i][1] = -(MOONOFFSET_Y*scale/ANIM)/2.5;
      v_point[i][2] = -(MOONOFFSET_Z*scale/ANIM)/4;
      for (int j = 0 ; j < 3 ; j++){
        h_buff[i][j] = 0;
      }
    }
    */

    st_point[i]=visc;
    e_point[i]=ref;
    J_point[i]=cap*M*temp;
    for (int j = 0; j < NUM_POINTS; j++) {
      colsyncindex[i][j]=NUM_POINTS;
    }
  }
  checkCudaErrors(hipMalloc((void**)&d_point, 3 * NUM_POINTS * sizeof(float)));
  checkCudaErrors(hipMalloc((void**)&dv_point, 3 * NUM_POINTS * sizeof(float)));
  checkCudaErrors(hipMalloc((void**)&v_buff, 3 * NUM_POINTS * sizeof(float)));
  checkCudaErrors(hipMalloc((void**)&dst_point, NUM_POINTS * sizeof(float)));
  checkCudaErrors(hipMalloc((void**)&de_point, NUM_POINTS * sizeof(float)));
  checkCudaErrors(hipMalloc((void**)&dJ_point, NUM_POINTS * sizeof(float)));
  checkCudaErrors(hipMalloc((void**)&dcolsynctime, NUM_POINTS*NUM_POINTS * sizeof(float)));
  checkCudaErrors(hipMalloc((void**)&dcolsyncindex, NUM_POINTS*NUM_POINTS * sizeof(int)));
  checkCudaErrors(hipMemcpy(d_point, h_point, 3 * NUM_POINTS * sizeof(float) , hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(dv_point, v_point, 3 * NUM_POINTS * sizeof(float) , hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(v_buff, h_buff, 3 * NUM_POINTS * sizeof(float) , hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(dst_point, st_point, NUM_POINTS * sizeof(float) , hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(de_point, e_point, NUM_POINTS * sizeof(float) , hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(dJ_point, J_point, NUM_POINTS * sizeof(float) , hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(dcolsynctime, colsynctime, NUM_POINTS*NUM_POINTS * sizeof(float) , hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(dcolsyncindex, colsyncindex, NUM_POINTS*NUM_POINTS * sizeof(int) , hipMemcpyHostToDevice));
}
void launchGPUKernel(unsigned int num_particles,float(*pos)[3],float(*vec)[3],float(*v_buff)[3],float(*sti),float(*e),float(*J),float(*coltime)[NUM_POINTS],int(*colindex)[NUM_POINTS])
{
    dim3 grid(Grid_x,Grid_y,Grid_z);
    dim3 block(Block_x,Block_y,Block_z);
    grav_coldetect<<<grid , block>>>(pos, vec,coltime,colindex);
    grav_colv<<<grid , block>>>(pos,vec,v_buff,sti,e,J,coltime,colindex);
    grav_v<<<grid , block>>>(pos,vec,v_buff,colindex);
    grav_vupdate<<<grid , block>>>(vec,v_buff);
    buff_clear<<<grid , block>>>(v_buff,coltime,colindex);
    grav_p<<<grid , block>>>(pos,vec);
}
void runGPUKernel(void)
{
  launchGPUKernel(NUM_POINTS, d_point, dv_point,v_buff,dst_point, de_point,dJ_point,dcolsynctime,dcolsyncindex);
  checkCudaErrors(hipMemcpy(h_point, d_point, 3 * NUM_POINTS * sizeof(float) , hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(v_point, dv_point, 3 * NUM_POINTS * sizeof(float) , hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(h_buff, v_buff, 3 * NUM_POINTS * sizeof(float) , hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(st_point, dst_point, NUM_POINTS * sizeof(float) , hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(e_point, de_point, NUM_POINTS * sizeof(float) , hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(J_point, dJ_point, NUM_POINTS * sizeof(float) , hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(colsynctime,dcolsynctime, NUM_POINTS*NUM_POINTS * sizeof(float) , hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(colsyncindex,dcolsyncindex, NUM_POINTS*NUM_POINTS * sizeof(int) , hipMemcpyDeviceToHost));
  anim_time += anim_dt;
}
void defineViewMatrix(double phi, double theta)
{
  unsigned int i;
  double c, s, xy_dist;
  double x_axis[3], y_axis[3], z_axis[3];

  eye[Z] = sin(theta * PI / 180.0);
  xy_dist = cos(theta * PI / 180.0);
  c = cos(phi * PI / 180.0);
  s = sin(phi * PI / 180.0);
  eye[X] = xy_dist * c;
  eye[Y] = xy_dist * s;
  up[X] = - c * eye[Z];
  up[Y] = - s * eye[Z];
  up[Z] = s * eye[Y] + c * eye[X];
  normVec(up);
  for (i = 0; i < 3; i++)
    {
      z_axis[i] = eye[i] - center[i];
    }
  normVec(z_axis);
  cross(up, z_axis, x_axis);
  normVec(x_axis);
  cross(z_axis, x_axis, y_axis);
  gluLookAt(eye[X], eye[Y], eye[Z], center[X], center[Y], center[Z], up[X], up[Y], up[Z]);
}
//円を描き、視点に合わせて向きを変えることで球を表現
void metaball (float pos[3], float color[3]) {
  double margin=0;
  double view[3]={0};
  double TH=theta;
  double PH=-phi;
  for (int i = 0 ; i < dev_points ; i ++)
    {
      view[0] = 0;
      view[1] = R * cos(i * PI * 2 / dev);
      view[2] = R * sin(i * PI * 2 / dev);
      point[i][X] = view[0] * cos(TH * PI / 180) * cos(PH * PI / 180) + view[1] * sin(PH * PI / 180) - view[2] * sin(TH * PI / 180) * cos(PH * PI / 180);
      point[i][Y] = - view[0] * cos(TH * PI / 180) * sin(PH * PI / 180) + view[1] * cos(PH * PI / 180) + view[2] * sin(TH * PI / 180) * sin(PH * PI / 180);
      point[i][Z] = view[0] * sin(TH * PI / 180) + view[2] * cos(TH * PI / 180);
    }
  glBegin(GL_TRIANGLE_FAN);
  glColor4f(1,1,1,0.3);
  glVertex3d(pos[X],pos[Y],pos[Z]);
  for (int i = 0 ; i < dev_points ; i ++)
    {
      glVertex3d(point[i][X] + pos[X], point[i][Y] + pos[Y], point[i][Z] + pos[Z]);
    }
  glEnd();
  glBegin(GL_POINTS);
  glColor4f(color[0],color[1],color[2],0.1);
  for (int k = 0; k < hollow; k++) {
    margin=(colmargin-1)/10*k+1;
    for (int i = 0 ; i < dev_points ; i ++)
      {
        if ((rand() % dev) < (dev / 2 / (k + 1)))
          {
            glVertex3d(margin*point[i][X] + pos[X], margin*point[i][Y] + pos[Y], margin*point[i][Z] + pos[Z]);
          }
      }
  }
  glEnd();
}
void display(void)
{
    light_pos[0] = (float)eye[X];
    light_pos[1] = (float)eye[Y];
    light_pos[2] = (float)eye[Z];
    light_pos[3] = 0.0f;
    runGPUKernel();
    glLightfv(GL_LIGHT0, GL_POSITION, light_pos);
    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    glOrtho(-vision_size-right_motion/2, vision_size+right_motion/2, -vision_size-right_motion/2, vision_size+right_motion/2, -100*vision_size, 100*vision_size);
    glViewport(0, 0, window_width, window_height);
    defineViewMatrix(phi, theta);
    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
    float color[3]={0};
    for (int k = 0 ; k < NUM_POINTS ; k++)
      {
        if(J_point[k]/M/cap-temp < resol){
          color[0]=1.0;
          color[1]=1.0;
          color[2]=1.0-(J_point[k]/M/cap-temp)/resol;
        }
        else if(J_point[k]/M/cap-temp < 2 * resol){
          color[0]=1.0;
          color[1]=1.0-(J_point[k]/M/cap-temp-resol)/resol;
          color[2]=0.0;
        }
        else {
          color[0]=1.0;
          color[1]=0.0;
          color[2]=0.0;
        }
        metaball(h_point[k],color);
      }
    glutSwapBuffers();
    glutPostRedisplay();
}

void mouse_button(int button, int state, int x, int y)
{
  if ((state == GLUT_DOWN) && (button == GLUT_LEFT_BUTTON))
    motion_p = true;
  if ((state == GLUT_DOWN) && (button == GLUT_RIGHT_BUTTON))
    motion_w = true;
  else if (state == GLUT_UP) {
    motion_p = false;
    motion_w = false;
  }
  mouse_old_x = x;
  mouse_old_y = y;
}
void mouse_motion(int x, int y)
{
  int dx, dy;
  dx = x - mouse_old_x;
  dy = y - mouse_old_y;
  if (motion_p) {
    phi -= dx * 0.2;
    theta += dy * 0.2;
  }
  if (motion_w) {
    right_motion += dx / 10;
    up_motion -= dy / 10;
  }
  mouse_old_x = x;
  mouse_old_y = y;
  glutPostRedisplay();
}

void resize(int width, int height)
{
  window_width = width;
  window_height = height;
}
void keyboard(unsigned char key, int x, int y)
{
  switch (key) {
  case 'q':
  case 'Q':
  case '\033':
    exit(0);

  default:
    break;
  }
}

bool initGL(void)
{
  glClearColor(0.0f, 0.0f , 0.0f, 0.5f);
  glEnable(GL_DEPTH_TEST);
  glClearDepth(1.0);
  glDepthFunc(GL_LESS);
  glEnable(GL_LIGHT0);
  return true;
}

int main(int argc, char** argv)
{
  point = (double **)malloc(sizeof(double *) * dev_points);
  for (int i = 0 ; i < dev_points ; i++)
    {
      point[i] = (double *)malloc(sizeof(double) * 3);
    }
  glutInit(&argc, argv);
  glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);
  glutInitWindowSize(window_width, window_height);
  glutCreateWindow("3D CUDA Simulation");
  glutDisplayFunc(display);
  glutReshapeFunc(resize);
  glutKeyboardFunc(keyboard);
  glutMouseFunc(mouse_button);
  glutMotionFunc(mouse_motion);
  setInitialPosition();
  if (!initGL())
    return 1;
  glutMainLoop();

  hipFree(dst_point);
  hipFree(de_point);
  hipFree(dJ_point);
  for (int i = 0 ; i < dev_points ; i++)
    {
      free (point[i]);
      hipFree(d_point[i]);
      hipFree(dv_point[i]);
      hipFree(v_buff[i]);
      hipFree(dcolsynctime[i]);
      hipFree(dcolsyncindex[i]);
    }
  free (point);
  hipFree(d_point);
  hipFree(dv_point);
  hipFree(v_buff);
  hipFree(dcolsynctime);
  hipFree(dcolsyncindex);
  hipDeviceReset();
  return 0;
}
