#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <GL/gl.h>
#include <GL/glut.h>
#include <math.h>
#include <stdbool.h>
#include <omp.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include <hip/hip_runtime.h>

//TDB
//���˗�p
//���^�{�[��
//VBO
//�������W�����`�ɂ���
//�Փˊp�x�⋗�����X���C�h�ŕύX�ł���悤�ɂ���

#define PI 3.141592653589793
//�����p�����[�^
#define cap 1000
#define ref 0.6
#define temp 4273
#define visc 9
#define GRAV (6.674*0.00000000000000000001)
#define density (2.5 * 1000000000000)
//���q�`��
#define rad 40 //km
#define M (4 / 3 * PI * rad*rad*rad* density)//kg
//�`�ʐݒ�
#define dev 30
#define resol 30
#define X 0
#define Y 1
#define Z 2
#define ANIM_START 0
#define ANIM 500
#define scale 0.01
#define colmargin 1.1
#define R (rad * scale)
#define INIT_WIDTH 800
#define INIT_HEIGHT 800
#define vision 40
#define Grid_x 16//block�Ԃ�__syncthread�ł͓����s��
#define Grid_y 8
#define Grid_z 1
#define Block_x 4
#define Block_y 2
#define Block_z 1

#define NUM_POINTS (Grid_x*Grid_y*Grid_z*Block_x*Block_y*Block_z)

unsigned int num_points = (dev + 1) * (dev + 1);
unsigned int window_width = INIT_WIDTH;
unsigned int window_height = INIT_HEIGHT;
double vision_size = vision;
float right_motion=0;
float up_motion=0;
double left, right, bottom, top;
float h_point[NUM_POINTS][3];
float v_point[NUM_POINTS][3];
float st_point[NUM_POINTS];
float e_point[NUM_POINTS];
float J_point[NUM_POINTS];
float h_buff[NUM_POINTS][3]={0};
float anim_time = ANIM_START;
float anim_dt = ANIM;
double phi = 30.0;
double theta = 30.0;
float light_pos[4];
int mouse_old_x, mouse_old_y;
bool motion_p;
bool motion_w;
double eye[3];
double center[3] = {0.0, 0.0, 0.0};
double up[3];
double ** point;
float (*d_point)[3];
float (*dv_point)[3];
float (*dst_point);
float (*de_point);
float (*dJ_point);
float (*v_buff)[3];
float colsynctime[NUM_POINTS][NUM_POINTS]={0};
int colsyncindex[NUM_POINTS][NUM_POINTS]={0};
float (*dcolsynctime)[NUM_POINTS];
int (*dcolsyncindex)[NUM_POINTS];
__global__ void grav_coldetect(float(*pos)[3],float(*vec)[3],float(*coltime)[NUM_POINTS],int(*colindex)[NUM_POINTS]);
__global__ void grav_colv(float(*pos)[3],float(*vec)[3],float(*v_buff)[3],float(*sti),float(*e),float(*J),float(*coltime)[NUM_POINTS],int(*colindex)[NUM_POINTS]);
__global__ void grav_v(float(*pos)[3],float(*vec)[3],float(*v_buff)[3],int(*colindex)[NUM_POINTS]);
__global__ void grav_vupdate(float(*vec)[3],float(*v_buff)[3]);
__global__ void buff_clear(float(*v_buff)[3],float(*coltime)[NUM_POINTS],int(*colindex)[NUM_POINTS]);
__global__ void grav_p(float (*pos)[3], float(*vec)[3]);

//��{�֐��Q
double dot(double vec0[], double vec1[])
{
  return(vec0[X] * vec1[X] + vec0[Y] * vec1[Y] + vec0[Z] * vec1[Z]);
}
void cross(double vec0[], double vec1[], double vec2[])
{
  vec2[X] = vec0[Y] * vec1[Z] - vec0[Z] * vec1[Y];
  vec2[Y] = vec0[Z] * vec1[X] - vec0[X] * vec1[Z];
  vec2[Z] = vec0[X] * vec1[Y] - vec0[Y] * vec1[X];
}
void normVec(double vec[])
{
  double norm;
  norm = sqrt(vec[X] * vec[X] + vec[Y] * vec[Y] + vec[Z] * vec[Z]);
  vec[X] /= norm;
  vec[Y] /= norm;
  vec[Z] /= norm;
}
void normal(double p0[], double p1[], double p2[], double normal[])
{
  unsigned int i;
  double v0[3], v1[3];
  for (i = 0; i < 3; i++) {
    v0[i] = p2[i] - p1[i];
    v1[i] = p0[i] - p1[i];
  }
  cross(v0, v1, normal);
  normVec(normal);
}

//�Փˌ��m
__global__ void grav_coldetect(float(*pos)[3],float(*vec)[3],float(*coltime)[NUM_POINTS],int(*colindex)[NUM_POINTS])
{
  float xn,yn,zn,vx,vy,vz,dis,sq;
  unsigned int thread_idx = threadIdx.x+blockDim.x*blockIdx.x;
  unsigned int thread_idy = threadIdx.y+blockDim.y*blockIdx.y;
  unsigned int thread_idz = threadIdx.z+blockDim.z*blockIdx.z;
  unsigned int index = (blockDim.x * Grid_x) * (blockDim.y * Grid_y) * thread_idz + (blockDim.x * Grid_x) * thread_idy + thread_idx ;
  float rvec[3]={0};
  xn = pos[index][0];
  yn = pos[index][1];
  zn = pos[index][2];
  vx = vec[index][0];
  vy = vec[index][1];
  vz = vec[index][2];
  for (int i = 0 ; i < NUM_POINTS; i++)
    {
      sq = (float)pow((double)(xn-pos[i][0]),2) + pow((double)(yn-pos[i][1]),2) + pow((double)(zn-pos[i][2]),2);
      dis = (float)sqrt((double)sq);
      rvec[0]=(pos[i][0]-xn)/dis;
      rvec[1]=(pos[i][1]-yn)/dis;
      rvec[2]=(pos[i][2]-zn)/dis;
      //�Փˈ�N������
      if (dis > 2 * R * colmargin  && i != index)
	{
	  colindex[index][i]=NUM_POINTS;
	}
      else if (dis <= 2 * R * colmargin && i != index)
	{
	  //�Փˈ�N������̌o�߂̎��Ԃ��L�^�@
	  colindex[index][i]=i;
	  coltime[index][i]=(2*R*colmargin-dis)/((vx-vec[i][0])*rvec[0]+(vy-vec[i][1])*rvec[1]+(vz-vec[i][2])*rvec[2]);
	}
      else
	{
	  colindex[index][i]=NUM_POINTS;
	}
    }
}

//�Փˌ�̑��x���v�Z
__global__ void grav_colv(float(*pos)[3],float(*vec)[3],float(*v_buff)[3],float(*sti),float(*e),float(*J),float(*coltime)[NUM_POINTS],int(*colindex)[NUM_POINTS])
{
  float xn,yn,zn,sq,dis;
  unsigned int thread_idx = threadIdx.x+blockDim.x*blockIdx.x;
  unsigned int thread_idy = threadIdx.y+blockDim.y*blockIdx.y;
  unsigned int thread_idz = threadIdx.z+blockDim.z*blockIdx.z;
  unsigned int index = (blockDim.x * Grid_x) * (blockDim.y * Grid_y) * thread_idz + (blockDim.x * Grid_x) * thread_idy + thread_idx ;
  int colnum=0;
  float tmptime=0;
  int tmpindex=0;
  int coldex=0;
  float repul=0;
  float rvec[3]={0};
  float Vl[3]={0};
  float Vr[3]={0};
  float Vh[3]={0};
  float vl_buff[3]={0};
  float vr_buff[3]={0};
  float vcol_buff[3]={0};
  float dotV=0;
  xn = pos[index][0];
  yn = pos[index][1];
  zn = pos[index][2];
  vl_buff[0]=vec[index][0];
  vl_buff[1]=vec[index][1];
  vl_buff[2]=vec[index][2];
  for (int i = 0 ; i < NUM_POINTS; i++){
    if(colindex[index][i]!=NUM_POINTS){
      colnum++;
    }
  }
  if(colnum>0){
      //�Փˈ�N������̌o�ߎ��Ԃ��C���f�b�N�X�t���\�[�g
      for (int i = 0 ; i < NUM_POINTS; i++){
        for(int j = i+1; j < NUM_POINTS; j++){
          if(coltime[index][i] > coltime[index][j]){
            tmptime=coltime[index][i];
            tmpindex=colindex[index][i];
            coltime[index][i]=coltime[index][j];
            colindex[index][i]=colindex[index][j];
            coltime[index][j]=tmptime;
            colindex[index][j]=tmpindex;
          }
        }
      }
      //�Փˈ�N������̌o�ߎ��Ԃ��������̂��珈��
      for (int i=NUM_POINTS-1 ; i>=NUM_POINTS-colnum; i--){
	coldex=colindex[index][i];
	sq = (float)pow((double)(xn-pos[coldex][0]),2)+pow((double)(yn-pos[coldex][1]),2)+pow((double)(zn-pos[coldex][2]),2);
	dis = (float)sqrt((double)sq);
	//�Փ˂̉^���ʂ̒P�ʃx�N�g��
	rvec[0]=(pos[coldex][0]-xn)/dis;
	rvec[1]=(pos[coldex][1]-yn)/dis;
	rvec[2]=(pos[coldex][2]-zn)/dis;
	//�����̑��x�x�N�g���̖@������
	dotV=rvec[0]*vl_buff[0]+rvec[1]*vl_buff[1]+rvec[2]*vl_buff[2];
        Vl[0]=dotV*rvec[0];
        Vl[1]=dotV*rvec[1];
        Vl[2]=dotV*rvec[2];
	//����̑��x�x�N�g���̖@������
	dotV=rvec[0]*vec[coldex][0]+rvec[1]*vec[coldex][1]+rvec[2]*vec[coldex][2];
	Vr[0]=dotV*rvec[0];
	Vr[1]=dotV*rvec[1];
	Vr[2]=dotV*rvec[2];
	//�����̑��x�x�N�g���̐�������
	Vh[0]=vl_buff[0]-Vl[0];
	Vh[1]=vl_buff[1]-Vl[1];
	Vh[2]=vl_buff[2]-Vl[2];
	//�����W���͏������ق���D��
	repul=e[index];
	if (e[coldex] < e[index]) {
	  repul=e[coldex];
	}
	//���x�X�V 
        vcol_buff[0]=Vh[0]+((1+repul)*Vr[0]+(1-repul)*Vl[0])/2;
        vcol_buff[1]=Vh[1]+((1+repul)*Vr[1]+(1-repul)*Vl[1])/2;
        vcol_buff[2]=Vh[2]+((1+repul)*Vr[2]+(1-repul)*Vl[2])/2;
	//����̑��x�v�Z
        vr_buff[0]=vec[coldex][0]-Vr[0]+((1+repul)*Vl[0]+(1-repul)*Vr[0])/2;
        vr_buff[1]=vec[coldex][1]-Vr[1]+((1+repul)*Vl[1]+(1-repul)*Vr[1])/2;
        vr_buff[2]=vec[coldex][2]-Vr[2]+((1+repul)*Vl[2]+(1-repul)*Vr[2])/2;
	//�Փ˃G�l���M�[��S���̔�ŕ��z���M�G�l���M�[�ϊ� TBD ���˗�p
	double Energy=0.5*M*(pow(vec[coldex][0],2)+pow(vec[coldex][1],2)+pow(vec[coldex][2],2)+pow(vl_buff[0],2)+pow(vl_buff[1],2)+pow(vl_buff[2],2) - (pow(vcol_buff[0],2)+pow(vcol_buff[1],2)+pow(vcol_buff[2],2)+pow(vr_buff[0],2)+pow(vr_buff[1],2)+pow(vr_buff[2],2))) / pow(scale,2) * 1000000;
	J[index] += Energy / (pow(10.0,(double)(sti[index]-sti[coldex]))+1);
	//���x���15000�x�Ƃ���
	if (J[index] > M * cap * 15000){
	  J[index] = M * cap * 15000;
	}
        vl_buff[0]=vcol_buff[0];
        vl_buff[1]=vcol_buff[1];
        vl_buff[2]=vcol_buff[2];
	//�S���Ɣ����W���̍X�V �����W���͉��x�㏸�ɑ΂����`�ɍ~���A�S����100�x�㏸��1���~��
	e[index] = 1 - ((1-ref)/temp * J[index]/M/cap);
	if ( e[index] < 0 ){ e[index] = 0; }
	if ( e[index] > 1 ){ e[index] = 1; }
	sti[index] = visc - ((J[index]/M/cap - temp) / 100);   
      }
      v_buff[index][0]=vl_buff[0];
      v_buff[index][1]=vl_buff[1];
      v_buff[index][2]=vl_buff[2];
  }
}
//�d�͉e����̑��x���v�Z
__global__ void grav_v(float(*pos)[3],float(*vec)[3],float(*v_buff)[3],int(*colindex)[NUM_POINTS])
{
  float xn,yn,zn,vx,vy,vz,sq,dis;
  unsigned int thread_idx = threadIdx.x+blockDim.x*blockIdx.x;
  unsigned int thread_idy = threadIdx.y+blockDim.y*blockIdx.y;
  unsigned int thread_idz = threadIdx.z+blockDim.z*blockIdx.z;
  unsigned int index = (blockDim.x * Grid_x) * (blockDim.y * Grid_y) * thread_idz + (blockDim.x * Grid_x) * thread_idy + thread_idx ;
  int colnum=0;
  float gravity=0;
  xn = pos[index][0];
  yn = pos[index][1];
  zn = pos[index][2];
  for (int i = 0 ; i < NUM_POINTS; i++){
    if(colindex[index][i]!=NUM_POINTS){
      colnum++;
    }
  }
  if(colnum==0){
    //�Փ˂Ȃ��Ȃ玩���ȊO�̂��ׂĂ���d�͉e�����󂯂�
    vx = vec[index][0];
    vy = vec[index][1];
    vz = vec[index][2]; 
    for (int i = 0 ; i < NUM_POINTS; i++){
      if (i!=index) {
	sq = (float)pow((double)(xn-pos[i][0]),2) + pow((double)(yn-pos[i][1]),2) + pow((double)(zn-pos[i][2]),2);
	gravity=GRAV*M/sq*scale*scale;
	dis = (float)sqrt((double)sq);
	vx += ((pos[i][0]-xn)/dis)*gravity*ANIM*scale;
	vy += ((pos[i][1]-yn)/dis)*gravity*ANIM*scale;
	vz += ((pos[i][2]-zn)/dis)*gravity*ANIM*scale;
      }
    }
  }
  else {
    //�Փ˂���Ȃ玩���ƏՓˑΏۈȊO����d�͉e�����󂯂� 
    vx = v_buff[index][0];
    vy = v_buff[index][1];
    vz = v_buff[index][2];
  }
  v_buff[index][0] = vx;
  v_buff[index][1] = vy;
  v_buff[index][2] = vz;
}
__global__ void grav_vupdate(float(*vec)[3],float(*v_buff)[3])
{
  unsigned int thread_idx = threadIdx.x+blockDim.x*blockIdx.x;
  unsigned int thread_idy = threadIdx.y+blockDim.y*blockIdx.y;
  unsigned int thread_idz = threadIdx.z+blockDim.z*blockIdx.z;
  unsigned int index = (blockDim.x * Grid_x) * (blockDim.y * Grid_y) * thread_idz + (blockDim.x * Grid_x) * thread_idy + thread_idx ;
  vec[index][0]=v_buff[index][0];
  vec[index][1]=v_buff[index][1];
  vec[index][2]=v_buff[index][2];
}
//�o�b�t�@�ރN���A
__global__ void buff_clear(float(*v_buff)[3],float(*coltime)[NUM_POINTS],int(*colindex)[NUM_POINTS])
{
  unsigned int thread_idx = threadIdx.x+blockDim.x*blockIdx.x;
  unsigned int thread_idy = threadIdx.y+blockDim.y*blockIdx.y;
  unsigned int thread_idz = threadIdx.z+blockDim.z*blockIdx.z;
  unsigned int index = (blockDim.x * Grid_x) * (blockDim.y * Grid_y) * thread_idz + (blockDim.x * Grid_x) * thread_idy + thread_idx ;
  for (int i=0; i < 3; i++){
    v_buff[index][i]=0;
  }
  for (int i=0; i < NUM_POINTS; i++){
    coltime[index][i]=0;
    colindex[index][i]=NUM_POINTS;
  }
}
//�d�͉e����̍��W������
__global__ void grav_p(float(*pos)[3], float(*vec)[3])
{
  float xn,yn,zn,vx,vy,vz;
  unsigned int thread_idx = threadIdx.x+blockDim.x*blockIdx.x;
  unsigned int thread_idy = threadIdx.y+blockDim.y*blockIdx.y;
  unsigned int thread_idz = threadIdx.z+blockDim.z*blockIdx.z;
  unsigned int index = ( blockDim.x * (Grid_x - 1) + blockDim.x ) * ( blockDim.y * (Grid_y - 1) + blockDim.y ) * thread_idz + ( blockDim.x * (Grid_x - 1) + blockDim.x ) * thread_idy + thread_idx ;
  xn = pos[index][0];
  yn = pos[index][1];
  zn = pos[index][2];
  vx = vec[index][0];
  vy = vec[index][1];
  vz = vec[index][2];
  pos[index][0] = xn + vx * ANIM;
  pos[index][1] = yn + vy * ANIM;
  pos[index][2] = zn + vz * ANIM;
}

// ���q�������ʒu�ɔz�u�D
void setInitialPosition(void)
{
  for (int i = 0; i < NUM_POINTS; i++) {
    for (int j = 0 ; j < 3 ; j++){
      h_point[i][j] = (float)(rand()-rand()) / RAND_MAX * INIT_WIDTH/vision*2 ;
      v_point[i][j] = 0;
      h_buff[i][j] = 0;
    }
    st_point[i]=visc;
    e_point[i]=ref;
    J_point[i]=cap*M*temp;
    for (int j = 0; j < NUM_POINTS; j++) {
      colsyncindex[i][j]=NUM_POINTS;
    }
  }
  checkCudaErrors(hipMalloc((void**)&d_point, 3 * NUM_POINTS * sizeof(float)));
  checkCudaErrors(hipMalloc((void**)&dv_point, 3 * NUM_POINTS * sizeof(float)));
  checkCudaErrors(hipMalloc((void**)&v_buff, 3 * NUM_POINTS * sizeof(float)));
  checkCudaErrors(hipMalloc((void**)&dst_point, NUM_POINTS * sizeof(float)));
  checkCudaErrors(hipMalloc((void**)&de_point, NUM_POINTS * sizeof(float)));
  checkCudaErrors(hipMalloc((void**)&dJ_point, NUM_POINTS * sizeof(float)));
  checkCudaErrors(hipMalloc((void**)&dcolsynctime, NUM_POINTS*NUM_POINTS * sizeof(float)));
  checkCudaErrors(hipMalloc((void**)&dcolsyncindex, NUM_POINTS*NUM_POINTS * sizeof(int)));
  checkCudaErrors(hipMemcpy(d_point, h_point, 3 * NUM_POINTS * sizeof(float) , hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(dv_point, v_point, 3 * NUM_POINTS * sizeof(float) , hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(v_buff, h_buff, 3 * NUM_POINTS * sizeof(float) , hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(dst_point, st_point, NUM_POINTS * sizeof(float) , hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(de_point, e_point, NUM_POINTS * sizeof(float) , hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(dJ_point, J_point, NUM_POINTS * sizeof(float) , hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(dcolsynctime, colsynctime, NUM_POINTS*NUM_POINTS * sizeof(float) , hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(dcolsyncindex, colsyncindex, NUM_POINTS*NUM_POINTS * sizeof(int) , hipMemcpyHostToDevice));
}
//CUDA���s�֐�
void launchGPUKernel(unsigned int num_particles,float(*pos)[3],float(*vec)[3],float(*v_buff)[3],float(*sti),float(*e),float(*J),float(*coltime)[NUM_POINTS],int(*colindex)[NUM_POINTS])
{
    dim3 grid(Grid_x,Grid_y,Grid_z);
    dim3 block(Block_x,Block_y,Block_z);
    grav_coldetect<<<grid , block>>>(pos, vec,coltime,colindex);
    grav_colv<<<grid , block>>>(pos,vec,v_buff,sti,e,J,coltime,colindex);
    grav_v<<<grid , block>>>(pos,vec,v_buff,colindex);
    grav_vupdate<<<grid , block>>>(vec,v_buff);
    buff_clear<<<grid , block>>>(v_buff,coltime,colindex);
    grav_p<<<grid , block>>>(pos,vec);
}
//�A�j���[�V��������
void runGPUKernel(void)
{
  launchGPUKernel(NUM_POINTS, d_point, dv_point,v_buff,dst_point, de_point,dJ_point,dcolsynctime,dcolsyncindex);
  checkCudaErrors(hipMemcpy(h_point, d_point, 3 * NUM_POINTS * sizeof(float) , hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(v_point, dv_point, 3 * NUM_POINTS * sizeof(float) , hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(h_buff, v_buff, 3 * NUM_POINTS * sizeof(float) , hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(st_point, dst_point, NUM_POINTS * sizeof(float) , hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(e_point, de_point, NUM_POINTS * sizeof(float) , hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(J_point, dJ_point, NUM_POINTS * sizeof(float) , hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(colsynctime,dcolsynctime, NUM_POINTS*NUM_POINTS * sizeof(float) , hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(colsyncindex,dcolsyncindex, NUM_POINTS*NUM_POINTS * sizeof(int) , hipMemcpyDeviceToHost));
  anim_time += anim_dt;
}
//�r���[��`
void defineViewMatrix(double phi, double theta)
{
  unsigned int i;
  double c, s, xy_dist;
  double x_axis[3], y_axis[3], z_axis[3];

  // ���_�̐ݒ�D
  eye[Z] = sin(theta * PI / 180.0);
  xy_dist = cos(theta * PI / 180.0);
  c = cos(phi * PI / 180.0);
  s = sin(phi * PI / 180.0);
  eye[X] = xy_dist * c;
  eye[Y] = xy_dist * s;
  up[X] = - c * eye[Z];
  up[Y] = - s * eye[Z];
  up[Z] = s * eye[Y] + c * eye[X];
  normVec(up);
  // ���_�����_�Ƃ�����W�n�̒�`�D
  for (i = 0; i < 3; i++)
    {
      z_axis[i] = eye[i] - center[i];
    }
  normVec(z_axis);
  cross(up, z_axis, x_axis);
  normVec(x_axis);
  cross(z_axis, x_axis, y_axis);
  gluLookAt(eye[X], eye[Y], eye[Z], center[X], center[Y], center[Z], up[X], up[Y], up[Z]); 
}

void metaball (float pos[3], float color[3]) {
  double nrml_vec[3];
  double margin=0;
  double view[3]={0};
  double TH=theta;
  double PH=-phi;
  for (int i = 0 ; i < dev + 1; i ++)
    {
      for (int j = 0 ; j < dev + 1; j++)
        {
	  view[0] = R * cos(j * PI * 2 / dev) * cos(i * PI * 2 / dev);
	  view[1] = R * sin(j * PI * 2 / dev) * cos(i * PI * 2 / dev);
          view[2] = R * sin(i * PI * 2 / dev);
	  //�ɍ��W�ϊ�
	  point[i * dev + j][X] = view[0] * cos(TH * PI / 180) * cos(PH * PI / 180) + view[1] * sin(PH * PI / 180) - view[2] * sin(TH * PI / 180) * cos(PH * PI / 180);
	  point[i * dev + j][Y] = - view[0] * cos(TH * PI / 180) * sin(PH * PI / 180) + view[1] * cos(PH * PI / 180) + view[2] * sin(TH * PI / 180) * sin(PH * PI / 180);
	  point[i * dev + j][Z] = view[0] * sin(TH * PI / 180) + view[2] * cos(TH * PI / 180);

	}
    }
  //���S�̋��� 
  glBegin(GL_QUADS);
  glColor4f(1,1,1,0.3);
  for (int i = 0 ; i < dev + 1 ; i ++)
    {
      for (int j = 0 ; j < 2 * dev + 1 ; j++)
	{
	  normal(point[i * (dev-1) + j],point[(i + 1) * (dev-1) + j + 1],point[(i+1) * (dev-1) + j],nrml_vec);
	  glNormal3dv(nrml_vec);
	  glVertex3d(point[i * (dev-1) + j][X] + pos[X], point[i * (dev-1) + j][Y] + pos[Y], point[i * (dev-1) + j][Z] + pos[Z]);
	  glVertex3d(point[(i + 1) * (dev-1) + j][X] + pos[X],point[(i + 1) * (dev-1) + j][Y] + pos[Y],point[(i + 1) * (dev-1) + j][Z] + pos[Z]);
	  glVertex3d(point[(i + 1) * (dev-1) + j + 1][X] + pos[X], point[(i + 1) * (dev-1) + j + 1][Y] + pos[Y], point[(i + 1) * (dev-1) + j + 1][Z] + pos[Z]);
	  glVertex3d(point[i * (dev-1) + j + 1][X] + pos[X],point[i * (dev-1) + j + 1][Y] + pos[Y],point[i * (dev-1) + j + 1][Z] + pos[Z]);
	}
    }
  glEnd(); 
  //���͂̃{��
  glBegin(GL_POINTS);
  glColor4f(color[0],color[1],color[2],0.1);
  for (int k = 0; k < 5; k++) {
    margin=(colmargin-1)*5/(k*2)+1;
    for (int i = 0 ; i < dev + 1 ; i ++)
      {
	for (int j = 0 ; j < 2 * dev + 1 ; j++)
	  {
	    normal(point[i * (dev-1) + j],point[(i + 1) * (dev-1) + j + 1],point[(i+1) * (dev-1) + j],nrml_vec);
	    glNormal3dv(nrml_vec);
	    glVertex3d(margin*point[i * (dev-1) + j][X] + pos[X], margin*point[i * (dev-1) + j][Y] + pos[Y], margin*point[i * (dev-1) + j][Z] + pos[Z]);
	  }
      }
  }
  glEnd();
}
void display(void)
{
    light_pos[0] = (float)eye[X];
    light_pos[1] = (float)eye[Y];
    light_pos[2] = (float)eye[Z];
    light_pos[3] = 0.0f;
    //CUDA�J�n
    runGPUKernel();
    // �����̐ݒ�
    glLightfv(GL_LIGHT0, GL_POSITION, light_pos);
    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    glOrtho(-vision_size-right_motion/2, vision_size+right_motion/2, -vision_size-right_motion/2, vision_size+right_motion/2, -100*vision_size, 100*vision_size);
    glViewport(0, 0, window_width, window_height);
    defineViewMatrix(phi, theta);
    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
    //���^�{�[��
    float color[3]={0};
    for (int k = 0 ; k < NUM_POINTS ; k++)
      {
	//���x�ɂ���ĐF��ω�
	if(J_point[k]/M/cap-temp < resol){ 
          color[0]=1.0;                                                                                                     
          color[1]=1.0;                                                                                                      
          color[2]=1.0-(J_point[k]/M/cap-temp)/resol;
        }
	else if(J_point[k]/M/cap-temp < 2 * resol){
          color[0]=1.0;
          color[1]=1.0-(J_point[k]/M/cap-temp-resol)/resol;
          color[2]=0.0;
        }
        else {
          color[0]=1.0;
          color[1]=0.0;
          color[2]=0.0;
        }
	metaball(h_point[k],color);
      }
    glutSwapBuffers();
    glutPostRedisplay();
}

void mouse_button(int button, int state, int x, int y)
{
  if ((state == GLUT_DOWN) && (button == GLUT_LEFT_BUTTON))
    motion_p = true;
  if ((state == GLUT_DOWN) && (button == GLUT_RIGHT_BUTTON))
    motion_w = true;
  else if (state == GLUT_UP) {
    motion_p = false;
    motion_w = false;
  }
  mouse_old_x = x;
  mouse_old_y = y;
}
void mouse_motion(int x, int y)
{
  int dx, dy;
  dx = x - mouse_old_x;
  dy = y - mouse_old_y;
  if (motion_p) {
    phi -= dx * 0.2;
    theta += dy * 0.2;
  }
  if (motion_w) {
    right_motion += dx / 10;
    up_motion -= dy / 10;
  }
  mouse_old_x = x;
  mouse_old_y = y;
  glutPostRedisplay();
}

void resize(int width, int height)
{
  window_width = width;
  window_height = height;
}
void keyboard(unsigned char key, int x, int y)
{
  switch (key) {
  case 'q':
  case 'Q':
  case '\033':
    exit(0);
    
  default:
    break;
  }
}

bool initGL(void)
{
  glClearColor(0.0f, 0.0f , 0.0f, 0.5f);
  glEnable(GL_DEPTH_TEST);
  glClearDepth(1.0);
  glDepthFunc(GL_LESS);
  glEnable(GL_LIGHT0);
  return true;
}

int main(int argc, char** argv)
{
  point = (double **)malloc(sizeof(double *) * num_points);
  for (int i = 0 ; i < num_points ; i++)
    {
      point[i] = (double *)malloc(sizeof(double) * 3);
    } 
    glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE | GLUT_DEPTH);
    glutInitWindowSize(window_width, window_height); 
    glutCreateWindow("3D CUDA Simulation");
    glutDisplayFunc(display);
    glutReshapeFunc(resize);
    glutKeyboardFunc(keyboard);
    glutMouseFunc(mouse_button);
    glutMotionFunc(mouse_motion);
    setInitialPosition();
    if (!initGL())
      return 1;
    glutMainLoop();

    hipFree(dst_point);
    hipFree(de_point);
    hipFree(dJ_point);
    for (int i = 0 ; i < num_points ; i++)
      {
       	free (point[i]);
	hipFree(d_point[i]);
        hipFree(dv_point[i]);
        hipFree(v_buff[i]);
	hipFree(dcolsynctime[i]);
	hipFree(dcolsyncindex[i]);
      }
    free (point);
    hipFree(d_point);
    hipFree(dv_point);
    hipFree(v_buff);
    hipFree(dcolsynctime);
    hipFree(dcolsyncindex);
    hipDeviceReset();
    return 0;
}
