#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include <time.h>
#include <math.h>

#define Grid_x 1024
#define Grid_y 1024
#define Grid_z 1
#define Block_x 16
#define Block_y 16
#define Block_z 1


__global__ void thread_num(unsigned int  *device_result);

/* timer */
int timer(void){
  time_t now = time(NULL);
  struct tm *pnow = localtime(&now);
  char buff[128]="";
  sprintf(buff,"%d:%d:%d",pnow->tm_hour,pnow->tm_min,pnow->tm_sec);
  printf("%s\n",buff);
  return 0;
}

int main(int argc, char** argv){

  FILE *outputfile;
  unsigned int num=0;
  unsigned int cycle=0;

  num=atoi(argv[1]);

  outputfile = fopen("./prime_num.txt", "w");
  if (outputfile == NULL) {
    printf("cannot open\n");
    exit(1);
  }
  timer();

  /*ブロックサイズとグリッドサイズの設定*/
  dim3 grid(Grid_x,Grid_y,Grid_z);
  dim3 block(Block_x,Block_y,Block_z);

  /*ホスト側の変数設定*/
  unsigned int  thread_size = (Grid_x * Grid_y * Grid_z) * (Block_x * Block_y * Block_z);
  unsigned int  *host_result;
  host_result = (unsigned int *)malloc(thread_size * sizeof(unsigned int));

  cycle = num / thread_size;

  /*デバイス側の変数設定*/
  unsigned int  *device_result;
  
  /*デバイスメモリ領域の確保*/
  checkCudaErrors(hipMalloc((void**)&device_result, thread_size * sizeof(unsigned int)));
  
  /*タイマーを作成して計測開始*/
  hipEvent_t start;
  hipEvent_t stop;
  checkCudaErrors(hipEventCreate(&start));
  checkCudaErrors(hipEventCreate(&stop));
  checkCudaErrors(hipEventRecord(start, NULL));
  printf("Range : 0 - %u\n",thread_size);

  /*カーネルの起動*/
  thread_num<<<grid , block>>>(device_result);
  hipDeviceSynchronize();
  
  /*タイマーを停止し実行時間を表示*/
  checkCudaErrors(hipEventRecord(stop, NULL));
  checkCudaErrors(hipEventSynchronize(stop));
  float msecTotal = 0.0f;
  checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));
  printf("Processing time: %f (msec)\n", msecTotal);
  
  /*再度タイマー開始*/
  checkCudaErrors(hipEventRecord(start, NULL));
  
  /*結果の領域確保とデバイス側からのメモリ転送*/
  host_result = (unsigned int *)malloc(thread_size * sizeof(unsigned int));
  checkCudaErrors(hipMemcpy(host_result, device_result, thread_size * sizeof(unsigned int ) , hipMemcpyDeviceToHost));
  
  /*タイマーを停止し実行時間を表示*/
  checkCudaErrors(hipEventRecord(stop, NULL));
  checkCudaErrors(hipEventSynchronize(stop));
  checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));
  printf("Memory copy time: %f (msec)\n", msecTotal);
  printf("Now Writing...\n");
  for( unsigned int  l = 0; l < thread_size; l++ ){ 
    if (host_result[l] != 0){
      fprintf(outputfile,"%lu\n", host_result[l]);
    }
  }
  fclose(outputfile);
  
  /*ホスト・デバイスメモリの開放*/
  free(host_result);
  checkCudaErrors(hipFree(device_result));
  timer();
  
  /*終了処理*/
  hipDeviceReset();
  exit(0);
}

__global__ void thread_num(unsigned int  *device_result){
  /*スレッドIDの割り当て*/
  /* メモ　: 
     blockDim = block size , 
     threadIdx = 0~blockDim-1 , 
     blockIdx = 0~grid size-1 , 
     max thread = blockDim * max blockIdx + max threadIdx 
  */
  unsigned int  thread_idx = threadIdx.x+blockDim.x*blockIdx.x;
  unsigned int  thread_idy = threadIdx.y+blockDim.y*blockIdx.y;
  unsigned int  thread_idz = threadIdx.z+blockDim.z*blockIdx.z;
  unsigned int thread_id = ( blockDim.x * (Grid_x - 1) + blockDim.x ) * ( blockDim.y * (Grid_y - 1) + blockDim.y ) * thread_idz + ( blockDim.x * (Grid_x - 1) + blockDim.x ) * thread_idy + thread_idx ;
  unsigned int dev = 0 ;
  int flag = 0;

  /*素数判定*/
  if ( thread_id == 1 ){
    device_result[thread_id] = 0;
  }else if ( thread_id == 2 ){
    device_result[thread_id] = 2;
  }else if ( thread_id % 2 == 0 ){ 
    device_result[thread_id] = 0;
  }else{
    dev = 3;
    while ( ( dev * dev ) <= thread_id ){
      if ( thread_id % dev == 0 ){ 
	flag=1;
	break;
      }
      dev += 2;
    }
    if (flag == 0){
      device_result[thread_id] = thread_id;
    }else if (flag == 1){
      device_result[thread_id] = 0;
    }
  }
}
